#include "hip/hip_runtime.h"
#include "cuda_model.h"
#include "cuda_kernels.cu"
#include <random>
#include <fstream>
#include <spdlog/spdlog.h>

namespace msmartcompute {

CUDAModel::CUDAModel(const std::vector<int>& layerSizes)
    : layerSizes_(layerSizes),
      currentDevice_(0),
      mixedPrecision_(false),
      dropoutRate_(0.0f),
      batchNormMomentum_(0.9f),
      trainingMode_(true)
{
    // Initialize CUDA resources
    CUDA_CHECK(hipSetDevice(currentDevice_));
    CUDA_CHECK(hipStreamCreate(&stream_));
    CUDA_CHECK(hipblasCreate(&cublasHandle_));
    CUDA_CHECK(hipsolverDnCreate(&cusolverHandle_));

    // Set stream for cuBLAS and cuSOLVER
    CUDA_CHECK(hipblasSetStream(cublasHandle_, stream_));
    CUDA_CHECK(hipsolverSetStream(cusolverHandle_, stream_));

    // Initialize layers
    layers_.resize(layerSizes_.size() - 1);
    allocateDeviceMemory();
    initializeWeights();
    initializeBatchNorm();
}

CUDAModel::~CUDAModel() {
    freeDeviceMemory();
    CUDA_CHECK(hipblasDestroy(cublasHandle_));
    CUDA_CHECK(hipsolverDnDestroy(cusolverHandle_));
    CUDA_CHECK(hipStreamDestroy(stream_));
}

void CUDAModel::setDevice(int deviceId) {
    if (deviceId != currentDevice_) {
        currentDevice_ = deviceId;
        CUDA_CHECK(hipSetDevice(deviceId));
    }
}

void CUDAModel::enableMixedPrecision(bool enable) {
    mixedPrecision_ = enable;
}

void CUDAModel::setDropoutRate(float rate) {
    dropoutRate_ = rate;
}

void CUDAModel::setBatchNormMomentum(float momentum) {
    batchNormMomentum_ = momentum;
}

std::vector<float> CUDAModel::forward(const DataBatch& batch) {
    // Copy input to device
    thrust::device_vector<float> input(batch.inputs.begin(), batch.inputs.end());
    
    // Forward pass through layers
    for (size_t i = 0; i < layers_.size(); ++i) {
        Layer& layer = layers_[i];
        
        // Matrix multiplication
        float alpha = 1.0f;
        float beta = 0.0f;
        CUDA_CHECK(hipblasSgemm(cublasHandle_, HIPBLAS_OP_N, HIPBLAS_OP_N,
            layerSizes_[i + 1], batch.size, layerSizes_[i],
            &alpha, layer.weights.data().get(), layerSizes_[i + 1],
            input.data().get(), layerSizes_[i],
            &beta, layer.activations.data().get(), layerSizes_[i + 1]));

        // Add bias
        thrust::transform(layer.activations.begin(), layer.activations.end(),
            layer.biases.begin(), layer.activations.begin(),
            thrust::plus<float>());

        // Apply batch normalization
        if (trainingMode_) {
            applyBatchNorm(layer, true);
        }

        // Apply activation
        applyActivation(layer, "relu");

        // Apply dropout
        if (trainingMode_ && dropoutRate_ > 0.0f) {
            applyDropout(layer.activations);
        }

        // Update input for next layer
        input = layer.activations;
    }

    // Copy result back to host
    std::vector<float> result(input.size());
    thrust::copy(input.begin(), input.end(), result.begin());
    return result;
}

void CUDAModel::backward(float loss) {
    // Compute output gradient
    thrust::device_vector<float> outputGrad(layers_.back().activations.size());
    thrust::fill(outputGrad.begin(), outputGrad.end(), loss);

    // Backward pass through layers
    for (int i = layers_.size() - 1; i >= 0; --i) {
        Layer& layer = layers_[i];
        
        // Compute activation gradient
        computeActivationGrad(layer, "relu");

        // Compute weight gradients
        computeGradients(outputGrad);

        // Update weights
        updateWeights(0.001f);  // Learning rate

        // Prepare gradient for next layer
        if (i > 0) {
            outputGrad = layer.gradients;
        }
    }
}

bool CUDAModel::save(const std::string& path) const {
    try {
        std::ofstream file(path, std::ios::binary);
        if (!file) {
            return false;
        }

        // Save layer sizes
        size_t numLayers = layerSizes_.size();
        file.write(reinterpret_cast<const char*>(&numLayers), sizeof(numLayers));
        file.write(reinterpret_cast<const char*>(layerSizes_.data()), 
                  numLayers * sizeof(int));

        // Save layer parameters
        for (const auto& layer : layers_) {
            // Save weights
            std::vector<float> weights(layer.weights.size());
            thrust::copy(layer.weights.begin(), layer.weights.end(), weights.begin());
            file.write(reinterpret_cast<const char*>(weights.data()), 
                      weights.size() * sizeof(float));

            // Save biases
            std::vector<float> biases(layer.biases.size());
            thrust::copy(layer.biases.begin(), layer.biases.end(), biases.begin());
            file.write(reinterpret_cast<const char*>(biases.data()), 
                      biases.size() * sizeof(float));

            // Save batch norm parameters
            std::vector<float> gamma(layer.gamma.size());
            thrust::copy(layer.gamma.begin(), layer.gamma.end(), gamma.begin());
            file.write(reinterpret_cast<const char*>(gamma.data()), 
                      gamma.size() * sizeof(float));

            std::vector<float> beta(layer.beta.size());
            thrust::copy(layer.beta.begin(), layer.beta.end(), beta.begin());
            file.write(reinterpret_cast<const char*>(beta.data()), 
                      beta.size() * sizeof(float));
        }

        return true;
    } catch (const std::exception& e) {
        spdlog::error("Failed to save model: {}", e.what());
        return false;
    }
}

bool CUDAModel::load(const std::string& path) {
    try {
        std::ifstream file(path, std::ios::binary);
        if (!file) {
            return false;
        }

        // Load layer sizes
        size_t numLayers;
        file.read(reinterpret_cast<char*>(&numLayers), sizeof(numLayers));
        layerSizes_.resize(numLayers);
        file.read(reinterpret_cast<char*>(layerSizes_.data()), 
                 numLayers * sizeof(int));

        // Reinitialize layers
        layers_.resize(layerSizes_.size() - 1);
        allocateDeviceMemory();

        // Load layer parameters
        for (auto& layer : layers_) {
            // Load weights
            std::vector<float> weights(layer.weights.size());
            file.read(reinterpret_cast<char*>(weights.data()), 
                     weights.size() * sizeof(float));
            thrust::copy(weights.begin(), weights.end(), layer.weights.begin());

            // Load biases
            std::vector<float> biases(layer.biases.size());
            file.read(reinterpret_cast<char*>(biases.data()), 
                     biases.size() * sizeof(float));
            thrust::copy(biases.begin(), biases.end(), layer.biases.begin());

            // Load batch norm parameters
            std::vector<float> gamma(layer.gamma.size());
            file.read(reinterpret_cast<char*>(gamma.data()), 
                     gamma.size() * sizeof(float));
            thrust::copy(gamma.begin(), gamma.end(), layer.gamma.begin());

            std::vector<float> beta(layer.beta.size());
            file.read(reinterpret_cast<char*>(beta.data()), 
                     beta.size() * sizeof(float));
            thrust::copy(beta.begin(), beta.end(), layer.beta.begin());
        }

        return true;
    } catch (const std::exception& e) {
        spdlog::error("Failed to load model: {}", e.what());
        return false;
    }
}

void CUDAModel::initializeWeights() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, 0.02f);

    for (auto& layer : layers_) {
        // Initialize weights
        std::vector<float> weights(layer.weights.size());
        for (auto& w : weights) {
            w = dist(gen);
        }
        thrust::copy(weights.begin(), weights.end(), layer.weights.begin());

        // Initialize biases
        std::vector<float> biases(layer.biases.size(), 0.0f);
        thrust::copy(biases.begin(), biases.end(), layer.biases.begin());

        // Initialize optimizer states
        thrust::fill(layer.momentum.begin(), layer.momentum.end(), 0.0f);
        thrust::fill(layer.velocity.begin(), layer.velocity.end(), 0.0f);
    }
}

void CUDAModel::initializeBatchNorm() {
    for (auto& layer : layers_) {
        // Initialize gamma (scale)
        thrust::fill(layer.gamma.begin(), layer.gamma.end(), 1.0f);
        
        // Initialize beta (shift)
        thrust::fill(layer.beta.begin(), layer.beta.end(), 0.0f);
        
        // Initialize running statistics
        thrust::fill(layer.runningMean.begin(), layer.runningMean.end(), 0.0f);
        thrust::fill(layer.runningVar.begin(), layer.runningVar.end(), 1.0f);
    }
}

void CUDAModel::computeGradients(const thrust::device_vector<float>& outputGrad) {
    for (auto& layer : layers_) {
        // Compute weight gradients using cuBLAS
        float alpha = 1.0f;
        float beta = 0.0f;
        CUDA_CHECK(hipblasSgemm(cublasHandle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
            layer.weights.size(), outputGrad.size(), layer.activations.size(),
            &alpha, layer.activations.data().get(), layer.weights.size(),
            outputGrad.data().get(), outputGrad.size(),
            &beta, layer.gradients.data().get(), layer.weights.size()));
    }
}

void CUDAModel::updateWeights(float learningRate) {
    for (auto& layer : layers_) {
        // Update weights using Adam optimizer
        int blockSize = 256;
        int numBlocks = (layer.weights.size() + blockSize - 1) / blockSize;
        
        adamUpdateKernel<<<numBlocks, blockSize, 0, stream_>>>(
            layer.weights.data().get(),
            layer.momentum.data().get(),
            layer.velocity.data().get(),
            layer.gradients.data().get(),
            layer.weights.size(),
            learningRate,
            0.9f,  // beta1
            0.999f,  // beta2
            1e-8f,  // epsilon
            1  // step
        );
    }
}

void CUDAModel::applyDropout(thrust::device_vector<float>& data) {
    int blockSize = 256;
    int numBlocks = (data.size() + blockSize - 1) / blockSize;
    
    dropoutKernel<<<numBlocks, blockSize, 0, stream_>>>(
        data.data().get(),
        dropoutMask_.data().get(),
        data.size(),
        dropoutRate_,
        randomStates_.data().get()
    );
}

void CUDAModel::applyBatchNorm(Layer& layer, bool training) {
    int blockSize = 256;
    int numBlocks = (layer.activations.size() + blockSize - 1) / blockSize;
    
    batchNormForwardKernel<<<numBlocks, blockSize, 0, stream_>>>(
        layer.activations.data().get(),
        layer.activations.data().get(),
        layer.gamma.data().get(),
        layer.beta.data().get(),
        layer.runningMean.data().get(),
        layer.runningVar.data().get(),
        1,  // batchSize
        layerSizes_[0],  // channels
        layer.activations.size() / layerSizes_[0],  // spatialSize
        batchNormMomentum_,
        1e-5f  // epsilon
    );
}

void CUDAModel::applyActivation(Layer& layer, const std::string& activationType) {
    int blockSize = 256;
    int numBlocks = (layer.activations.size() + blockSize - 1) / blockSize;
    
    if (activationType == "relu") {
        reluKernel<<<numBlocks, blockSize, 0, stream_>>>(
            layer.activations.data().get(),
            layer.activations.size()
        );
    }
}

void CUDAModel::computeActivationGrad(Layer& layer, const std::string& activationType) {
    int blockSize = 256;
    int numBlocks = (layer.activations.size() + blockSize - 1) / blockSize;
    
    if (activationType == "relu") {
        reluGradKernel<<<numBlocks, blockSize, 0, stream_>>>(
            layer.activations.data().get(),
            layer.gradients.data().get(),
            layer.gradients.data().get(),
            layer.activations.size()
        );
    }
}

void CUDAModel::allocateDeviceMemory() {
    for (size_t i = 0; i < layers_.size(); ++i) {
        Layer& layer = layers_[i];
        size_t inputSize = layerSizes_[i];
        size_t outputSize = layerSizes_[i + 1];
        
        // Allocate memory for weights and biases
        layer.weights.resize(inputSize * outputSize);
        layer.biases.resize(outputSize);
        layer.activations.resize(outputSize);
        layer.gradients.resize(inputSize * outputSize);
        layer.momentum.resize(inputSize * outputSize);
        layer.velocity.resize(inputSize * outputSize);
        
        // Allocate memory for batch normalization
        layer.gamma.resize(outputSize);
        layer.beta.resize(outputSize);
        layer.runningMean.resize(outputSize);
        layer.runningVar.resize(outputSize);
    }
    
    // Allocate memory for dropout
    dropoutMask_.resize(layerSizes_.back());
    randomStates_.resize(layerSizes_.back());
    
    // Initialize random states
    int blockSize = 256;
    int numBlocks = (randomStates_.size() + blockSize - 1) / blockSize;
    hiprand_init<<<numBlocks, blockSize, 0, stream_>>>(
        time(nullptr),
        thrust::raw_pointer_cast(randomStates_.data()),
        randomStates_.size()
    );
}

void CUDAModel::freeDeviceMemory() {
    for (auto& layer : layers_) {
        layer.weights.clear();
        layer.biases.clear();
        layer.activations.clear();
        layer.gradients.clear();
        layer.momentum.clear();
        layer.velocity.clear();
        layer.gamma.clear();
        layer.beta.clear();
        layer.runningMean.clear();
        layer.runningVar.clear();
    }
    dropoutMask_.clear();
    randomStates_.clear();
}

void CUDAModel::synchronizeDevice() {
    CUDA_CHECK(hipStreamSynchronize(stream_));
}

} // namespace msmartcompute 