#include "hip/hip_runtime.h"
#include "cuda_optimizer.h"
#include "cuda_kernels.cu"
#include <spdlog/spdlog.h>

namespace msmartcompute {

// CUDA Adam Optimizer Implementation
CUDAAdamOptimizer::CUDAAdamOptimizer(float learningRate, float beta1, float beta2, float epsilon)
    : learningRate_(learningRate),
      beta1_(beta1),
      beta2_(beta2),
      epsilon_(epsilon),
      step_(0)
{
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDAAdamOptimizer::~CUDAAdamOptimizer() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

void CUDAAdamOptimizer::step() {
    step_++;
    
    // Update weights using Adam optimizer
    int blockSize = 256;
    int numBlocks = (momentum_.size() + blockSize - 1) / blockSize;
    
    adamUpdateKernel<<<numBlocks, blockSize, 0, stream_>>>(
        thrust::raw_pointer_cast(momentum_.data()),
        thrust::raw_pointer_cast(velocity_.data()),
        thrust::raw_pointer_cast(momentum_.data()),
        thrust::raw_pointer_cast(velocity_.data()),
        momentum_.size(),
        learningRate_,
        beta1_,
        beta2_,
        epsilon_,
        step_
    );
}

float CUDAAdamOptimizer::getLearningRate() const {
    return learningRate_;
}

void CUDAAdamOptimizer::setLearningRate(float lr) {
    learningRate_ = lr;
}

void CUDAAdamOptimizer::setBeta1(float beta1) {
    beta1_ = beta1;
}

void CUDAAdamOptimizer::setBeta2(float beta2) {
    beta2_ = beta2;
}

void CUDAAdamOptimizer::setEpsilon(float epsilon) {
    epsilon_ = epsilon;
}

// CUDA SGD Optimizer Implementation
CUDASGDOptimizer::CUDASGDOptimizer(float learningRate, float momentum, float weightDecay)
    : learningRate_(learningRate),
      momentum_(momentum),
      weightDecay_(weightDecay)
{
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDASGDOptimizer::~CUDASGDOptimizer() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

void CUDASGDOptimizer::step() {
    int blockSize = 256;
    int numBlocks = (velocity_.size() + blockSize - 1) / blockSize;
    
    // Update velocity with momentum
    if (momentum_ > 0.0f) {
        thrust::transform(
            velocity_.begin(),
            velocity_.end(),
            velocity_.begin(),
            [this](float v) { return momentum_ * v; }
        );
    }
    
    // Add gradient to velocity
    thrust::transform(
        velocity_.begin(),
        velocity_.end(),
        velocity_.begin(),
        [this](float v) { return v - learningRate_ * v; }
    );
    
    // Apply weight decay if enabled
    if (weightDecay_ > 0.0f) {
        thrust::transform(
            velocity_.begin(),
            velocity_.end(),
            velocity_.begin(),
            [this](float v) { return v - learningRate_ * weightDecay_ * v; }
        );
    }
}

float CUDASGDOptimizer::getLearningRate() const {
    return learningRate_;
}

void CUDASGDOptimizer::setLearningRate(float lr) {
    learningRate_ = lr;
}

void CUDASGDOptimizer::setMomentum(float momentum) {
    momentum_ = momentum;
}

void CUDASGDOptimizer::setWeightDecay(float weightDecay) {
    weightDecay_ = weightDecay;
}

// CUDA RMSProp Optimizer Implementation
CUDARMSPropOptimizer::CUDARMSPropOptimizer(float learningRate, float alpha, float epsilon, float weightDecay)
    : learningRate_(learningRate),
      alpha_(alpha),
      epsilon_(epsilon),
      weightDecay_(weightDecay)
{
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDARMSPropOptimizer::~CUDARMSPropOptimizer() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

void CUDARMSPropOptimizer::step() {
    int blockSize = 256;
    int numBlocks = (squareAvg_.size() + blockSize - 1) / blockSize;
    
    // Update square average
    thrust::transform(
        squareAvg_.begin(),
        squareAvg_.end(),
        squareAvg_.begin(),
        [this](float avg) { return alpha_ * avg + (1.0f - alpha_) * avg * avg; }
    );
    
    // Update weights
    thrust::transform(
        squareAvg_.begin(),
        squareAvg_.end(),
        squareAvg_.begin(),
        [this](float avg) {
            float update = -learningRate_ / (std::sqrt(avg) + epsilon_);
            if (weightDecay_ > 0.0f) {
                update -= learningRate_ * weightDecay_;
            }
            return update;
        }
    );
}

float CUDARMSPropOptimizer::getLearningRate() const {
    return learningRate_;
}

void CUDARMSPropOptimizer::setLearningRate(float lr) {
    learningRate_ = lr;
}

void CUDARMSPropOptimizer::setAlpha(float alpha) {
    alpha_ = alpha;
}

void CUDARMSPropOptimizer::setEpsilon(float epsilon) {
    epsilon_ = epsilon;
}

void CUDARMSPropOptimizer::setWeightDecay(float weightDecay) {
    weightDecay_ = weightDecay;
}

} // namespace msmartcompute 