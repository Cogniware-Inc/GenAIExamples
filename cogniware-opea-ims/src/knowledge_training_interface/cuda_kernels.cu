#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipsolver.h>

namespace msmartcompute {
namespace cuda {

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Forward pass kernel
__global__ void forwardKernel(
    const float* input,
    const float* weights,
    float* output,
    int batchSize,
    int inputSize,
    int outputSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batchSize * outputSize) {
        int batch = idx / outputSize;
        int outIdx = idx % outputSize;
        float sum = 0.0f;
        
        for (int i = 0; i < inputSize; i++) {
            sum += input[batch * inputSize + i] * weights[i * outputSize + outIdx];
        }
        
        output[idx] = sum;
    }
}

// Backward pass kernel
__global__ void backwardKernel(
    const float* input,
    const float* gradOutput,
    float* gradWeights,
    float* gradInput,
    int batchSize,
    int inputSize,
    int outputSize,
    float learningRate
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < inputSize * outputSize) {
        int inIdx = idx / outputSize;
        int outIdx = idx % outputSize;
        float grad = 0.0f;
        
        for (int b = 0; b < batchSize; b++) {
            grad += input[b * inputSize + inIdx] * gradOutput[b * outputSize + outIdx];
        }
        
        gradWeights[idx] -= learningRate * grad;
    }
}

// Loss computation kernel
__global__ void computeLossKernel(
    const float* output,
    const float* target,
    float* loss,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float diff = output[idx] - target[idx];
        loss[idx] = 0.5f * diff * diff;  // MSE loss
    }
}

// Activation function kernels
__global__ void reluKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = max(0.0f, data[idx]);
    }
}

__global__ void reluGradKernel(
    const float* data,
    const float* grad,
    float* output,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = data[idx] > 0.0f ? grad[idx] : 0.0f;
    }
}

// Optimizer kernels
__global__ void adamUpdateKernel(
    float* weights,
    float* momentum,
    float* velocity,
    const float* gradients,
    int size,
    float learningRate,
    float beta1,
    float beta2,
    float epsilon,
    int step
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        momentum[idx] = beta1 * momentum[idx] + (1.0f - beta1) * gradients[idx];
        velocity[idx] = beta2 * velocity[idx] + (1.0f - beta2) * gradients[idx] * gradients[idx];
        
        float mHat = momentum[idx] / (1.0f - powf(beta1, step));
        float vHat = velocity[idx] / (1.0f - powf(beta2, step));
        
        weights[idx] -= learningRate * mHat / (sqrtf(vHat) + epsilon);
    }
}

// Batch normalization kernels
__global__ void batchNormForwardKernel(
    const float* input,
    float* output,
    const float* gamma,
    const float* beta,
    float* runningMean,
    float* runningVar,
    int batchSize,
    int channels,
    int spatialSize,
    float momentum,
    float epsilon
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batchSize * channels * spatialSize) {
        int c = (idx / spatialSize) % channels;
        float mean = runningMean[c];
        float var = runningVar[c];
        
        float normalized = (input[idx] - mean) / sqrtf(var + epsilon);
        output[idx] = gamma[c] * normalized + beta[c];
    }
}

// Attention mechanism kernels
__global__ void selfAttentionKernel(
    const float* query,
    const float* key,
    const float* value,
    float* output,
    int batchSize,
    int seqLen,
    int headSize,
    int numHeads
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batchSize * seqLen * headSize) {
        int b = idx / (seqLen * headSize);
        int s = (idx / headSize) % seqLen;
        int h = idx % headSize;
        
        float sum = 0.0f;
        for (int t = 0; t < seqLen; t++) {
            float score = 0.0f;
            for (int d = 0; d < headSize; d++) {
                score += query[b * seqLen * headSize + s * headSize + d] *
                        key[b * seqLen * headSize + t * headSize + d];
            }
            score /= sqrtf(headSize);
            
            for (int d = 0; d < headSize; d++) {
                sum += score * value[b * seqLen * headSize + t * headSize + d];
            }
        }
        output[idx] = sum;
    }
}

// Dropout kernel
__global__ void dropoutKernel(
    float* data,
    float* mask,
    int size,
    float dropoutRate,
    hiprandState* states
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float random = hiprand_uniform(&states[idx]);
        mask[idx] = random > dropoutRate ? 1.0f : 0.0f;
        data[idx] *= mask[idx];
    }
}

// Metric computation kernels
__global__ void computeAccuracyKernel(
    const float* output,
    const float* target,
    float* accuracy,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int pred = roundf(output[idx]);
        int actual = roundf(target[idx]);
        accuracy[idx] = (pred == actual) ? 1.0f : 0.0f;
    }
}

__global__ void computePrecisionKernel(
    const float* output,
    const float* target,
    float* precision,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int pred = roundf(output[idx]);
        int actual = roundf(target[idx]);
        precision[idx] = (pred == 1 && actual == 1) ? 1.0f : 0.0f;
    }
}

// CUDA stream and handle management
class CUDAManager {
public:
    static CUDAManager& getInstance() {
        static CUDAManager instance;
        return instance;
    }

    hipblasHandle_t getCublasHandle() { return cublasHandle_; }
    hipsolverHandle_t getCusolverHandle() { return cusolverHandle_; }
    hipStream_t getStream() { return stream_; }

private:
    CUDAManager() {
        CUDA_CHECK(hipblasCreate(&cublasHandle_));
        CUDA_CHECK(hipsolverDnCreate(&cusolverHandle_));
        CUDA_CHECK(hipStreamCreate(&stream_));
    }

    ~CUDAManager() {
        hipblasDestroy(cublasHandle_);
        hipsolverDnDestroy(cusolverHandle_);
        hipStreamDestroy(stream_);
    }

    hipblasHandle_t cublasHandle_;
    hipsolverHandle_t cusolverHandle_;
    hipStream_t stream_;
};

} // namespace cuda
} // namespace msmartcompute 