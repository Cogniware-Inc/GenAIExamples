#include "cuda_memory_manager.h"
#include <spdlog/spdlog.h>
#include <algorithm>
#include <cstring>

namespace msmartcompute {

CUDAMemoryManager& CUDAMemoryManager::getInstance() {
    static CUDAMemoryManager instance;
    return instance;
}

bool CUDAMemoryManager::initialize(const MemoryPoolConfig& config) {
    config_ = config;
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to set CUDA device: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    // Create memory pool
    if (config_.strategy == MemoryStrategy::POOL) {
        size_t poolSize = config_.initialPoolSize;
        cudaStatus = hipMalloc(&poolMemory_, poolSize);
        if (cudaStatus != hipSuccess) {
            spdlog::error("Failed to allocate memory pool: {}", hipGetErrorString(cudaStatus));
            return false;
        }
        
        // Initialize free blocks list
        MemoryBlock block;
        block.address = poolMemory_;
        block.size = poolSize;
        block.isFree = true;
        freeBlocks_.push_back(block);
    }
    
    return true;
}

void CUDAMemoryManager::shutdown() {
    // Free all allocated memory
    for (auto& block : allocatedBlocks_) {
        if (block.address) {
            hipFree(block.address);
        }
    }
    allocatedBlocks_.clear();
    
    // Free memory pool
    if (poolMemory_) {
        hipFree(poolMemory_);
        poolMemory_ = nullptr;
    }
    freeBlocks_.clear();
}

bool CUDAMemoryManager::setMemoryStrategy(MemoryStrategy strategy) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (strategy == strategy_) {
        return true;
    }

    if (strategy == MemoryStrategy::POOL) {
        if (!initializePool()) {
            spdlog::error("Failed to initialize memory pool");
            return false;
        }
    } else {
        cleanupPool();
    }

    strategy_ = strategy;
    return true;
}

MemoryStrategy CUDAMemoryManager::getMemoryStrategy() const {
    return strategy_;
}

void* CUDAMemoryManager::allocate(size_t size, size_t alignment) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (config_.strategy == MemoryStrategy::POOL) {
        return allocateFromPool(size, alignment);
    } else if (config_.strategy == MemoryStrategy::DIRECT) {
        return allocateDirect(size, alignment);
    } else { // STREAMING
        return allocateStreaming(size, alignment);
    }
}

void CUDAMemoryManager::free(void* ptr) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (!ptr) return;
    
    if (config_.strategy == MemoryStrategy::POOL) {
        freeToPool(ptr);
    } else {
        hipFree(ptr);
    }
}

void* CUDAMemoryManager::reallocate(void* ptr, size_t newSize) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (!ptr) {
        return allocate(newSize);
    }
    
    if (config_.strategy == MemoryStrategy::POOL) {
        return reallocateInPool(ptr, newSize);
    } else {
        void* newPtr = allocate(newSize);
        if (newPtr) {
            // Copy data to new location
            hipMemcpy(newPtr, ptr, newSize, hipMemcpyDeviceToDevice);
            free(ptr);
        }
        return newPtr;
    }
}

bool CUDAMemoryManager::copyToDevice(void* dst, const void* src, size_t size) {
    hipError_t status = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        spdlog::error("Failed to copy memory to device: {}", hipGetErrorString(status));
        return false;
    }
    return true;
}

bool CUDAMemoryManager::copyToHost(void* dst, const void* src, size_t size) {
    hipError_t status = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        spdlog::error("Failed to copy memory to host: {}", hipGetErrorString(status));
        return false;
    }
    return true;
}

bool CUDAMemoryManager::memset(void* ptr, int value, size_t size) {
    hipError_t status = hipMemset(ptr, value, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to set memory: {}", hipGetErrorString(status));
        return false;
    }
    return true;
}

size_t CUDAMemoryManager::getTotalMemory() const {
    return totalMemory_;
}

size_t CUDAMemoryManager::getFreeMemory() const {
    return freeMemory_;
}

size_t CUDAMemoryManager::getUsedMemory() const {
    return usedMemory_;
}

void CUDAMemoryManager::defragment() {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (config_.strategy != MemoryStrategy::POOL) return;
    
    // Sort free blocks by address
    std::sort(freeBlocks_.begin(), freeBlocks_.end(),
        [](const MemoryBlock& a, const MemoryBlock& b) {
            return a.address < b.address;
        });
    
    // Merge adjacent free blocks
    for (size_t i = 0; i < freeBlocks_.size() - 1; ++i) {
        if (freeBlocks_[i].isFree && freeBlocks_[i + 1].isFree) {
            uintptr_t endAddr = reinterpret_cast<uintptr_t>(freeBlocks_[i].address) + freeBlocks_[i].size;
            if (endAddr == reinterpret_cast<uintptr_t>(freeBlocks_[i + 1].address)) {
                freeBlocks_[i].size += freeBlocks_[i + 1].size;
                freeBlocks_.erase(freeBlocks_.begin() + i + 1);
                --i;
            }
        }
    }
}

void CUDAMemoryManager::clear() {
    std::lock_guard<std::mutex> lock(mutex_);
    
    // Free all allocated memory
    for (auto& block : allocatedBlocks_) {
        if (block.address) {
            hipFree(block.address);
        }
    }
    allocatedBlocks_.clear();
    
    // Reset memory pool
    if (poolMemory_) {
        hipFree(poolMemory_);
        poolMemory_ = nullptr;
        
        size_t poolSize = config_.initialPoolSize;
        hipMalloc(&poolMemory_, poolSize);
        
        MemoryBlock block;
        block.address = poolMemory_;
        block.size = poolSize;
        block.isFree = true;
        freeBlocks_.clear();
        freeBlocks_.push_back(block);
    }
}

void CUDAMemoryManager::setMemoryCallback(std::function<void(size_t, size_t)> callback) {
    std::lock_guard<std::mutex> lock(mutex_);
    memoryCallback_ = callback;
}

void CUDAMemoryManager::enableMemoryTracking(bool enable) {
    std::lock_guard<std::mutex> lock(mutex_);
    trackingEnabled_ = enable;
}

void CUDAMemoryManager::printMemoryStats() const {
    spdlog::info("Memory Stats:");
    spdlog::info("  Total Memory: {} bytes", totalMemory_);
    spdlog::info("  Used Memory: {} bytes", usedMemory_);
    spdlog::info("  Free Memory: {} bytes", freeMemory_);
    spdlog::info("  Memory Utilization: {:.2f}%", 
        (static_cast<float>(usedMemory_) / totalMemory_) * 100.0f);
}

bool CUDAMemoryManager::initializePool() {
    void* ptr = nullptr;
    hipError_t status = hipMalloc(&ptr, config_.initialSize);
    if (status != hipSuccess) {
        spdlog::error("Failed to allocate initial memory pool: {}", hipGetErrorString(status));
        return false;
    }

    MemoryBlock block{ptr, config_.initialSize, false, nullptr, "pool"};
    blocks_.push_back(block);

    totalMemory_ = config_.initialSize;
    usedMemory_ = 0;
    freeMemory_ = config_.initialSize;

    return true;
}

void CUDAMemoryManager::cleanupPool() {
    for (const auto& block : blocks_) {
        if (block.ptr) {
            hipFree(block.ptr);
        }
    }
    blocks_.clear();
    totalMemory_ = 0;
    usedMemory_ = 0;
    freeMemory_ = 0;
}

void* CUDAMemoryManager::allocateFromPool(size_t size, size_t alignment) {
    // Find best fit block
    auto bestFit = std::min_element(freeBlocks_.begin(), freeBlocks_.end(),
        [size](const MemoryBlock& a, const MemoryBlock& b) {
            if (!a.isFree) return false;
            if (!b.isFree) return true;
            return a.size >= size && (b.size < size || a.size < b.size);
        });
    
    if (bestFit == freeBlocks_.end() || bestFit->size < size) {
        // No suitable block found, try to defragment
        defragment();
        
        // Try again after defragmentation
        bestFit = std::min_element(freeBlocks_.begin(), freeBlocks_.end(),
            [size](const MemoryBlock& a, const MemoryBlock& b) {
                if (!a.isFree) return false;
                if (!b.isFree) return true;
                return a.size >= size && (b.size < size || a.size < b.size);
            });
            
        if (bestFit == freeBlocks_.end() || bestFit->size < size) {
            spdlog::error("Failed to allocate memory from pool: insufficient space");
            return nullptr;
        }
    }
    
    // Split block if necessary
    if (bestFit->size > size + config_.minBlockSize) {
        MemoryBlock newBlock;
        newBlock.address = static_cast<char*>(bestFit->address) + size;
        newBlock.size = bestFit->size - size;
        newBlock.isFree = true;
        
        bestFit->size = size;
        freeBlocks_.push_back(newBlock);
    }
    
    bestFit->isFree = false;
    allocatedBlocks_.push_back(*bestFit);
    
    return bestFit->address;
}

void* CUDAMemoryManager::allocateDirect(size_t size, size_t alignment) {
    void* ptr = nullptr;
    hipError_t status = hipMalloc(&ptr, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to allocate memory directly: {}", hipGetErrorString(status));
        return nullptr;
    }
    
    MemoryBlock block;
    block.address = ptr;
    block.size = size;
    block.isFree = false;
    allocatedBlocks_.push_back(block);
    
    return ptr;
}

void* CUDAMemoryManager::allocateStreaming(size_t size, size_t alignment) {
    // For streaming strategy, we use a circular buffer
    if (streamingBuffer_.size() < config_.numStreamingBuffers) {
        void* ptr = nullptr;
        hipError_t status = hipMalloc(&ptr, size);
        if (status != hipSuccess) {
            spdlog::error("Failed to allocate streaming buffer: {}", hipGetErrorString(status));
            return nullptr;
        }
        
        streamingBuffer_.push_back(ptr);
        return ptr;
    }
    
    // Reuse oldest buffer
    void* ptr = streamingBuffer_.front();
    streamingBuffer_.pop_front();
    streamingBuffer_.push_back(ptr);
    
    return ptr;
}

void CUDAMemoryManager::freeToPool(void* ptr) {
    auto it = std::find_if(allocatedBlocks_.begin(), allocatedBlocks_.end(),
        [ptr](const MemoryBlock& block) {
            return block.address == ptr;
        });
        
    if (it != allocatedBlocks_.end()) {
        it->isFree = true;
        freeBlocks_.push_back(*it);
        allocatedBlocks_.erase(it);
    }
}

void* CUDAMemoryManager::reallocateInPool(void* ptr, size_t newSize) {
    auto it = std::find_if(allocatedBlocks_.begin(), allocatedBlocks_.end(),
        [ptr](const MemoryBlock& block) {
            return block.address == ptr;
        });
        
    if (it == allocatedBlocks_.end()) {
        return nullptr;
    }
    
    if (it->size >= newSize) {
        return ptr;
    }
    
    void* newPtr = allocateFromPool(newSize, 0);
    if (newPtr) {
        hipMemcpy(newPtr, ptr, it->size, hipMemcpyDeviceToDevice);
        freeToPool(ptr);
    }
    
    return newPtr;
}

} // namespace msmartcompute 