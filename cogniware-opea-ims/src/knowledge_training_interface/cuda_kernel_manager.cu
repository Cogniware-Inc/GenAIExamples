#include "hip/hip_runtime.h"
#include "cuda_kernel_manager.h"
#include <spdlog/spdlog.h>

namespace msmartcompute {

CUDAKernelManager& CUDAKernelManager::getInstance() {
    static CUDAKernelManager instance;
    return instance;
}

bool CUDAKernelManager::initialize(const KernelConfig& config) {
    config_ = config;
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to set CUDA device: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    // Initialize cuDNN
    hipdnnStatus_t cudnnStatus = hipdnnCreate(&cudnnHandle_);
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuDNN handle: {}", hipdnnGetErrorString(cudnnStatus));
        return false;
    }
    
    // Initialize cuBLAS
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle_);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuBLAS handle: {}", hipblasStatus_t);
        return false;
    }
    
    // Create streams
    for (int i = 0; i < config_.numStreams; ++i) {
        hipStream_t stream;
        cudaStatus = hipStreamCreate(&stream);
        if (cudaStatus != hipSuccess) {
            spdlog::error("Failed to create CUDA stream: {}", hipGetErrorString(cudaStatus));
            return false;
        }
        streams_.push_back(stream);
    }
    
    // Enable tensor cores if supported
    if (config_.useTensorCores) {
        hipblasStatus_t = hipblasSetMathMode(cublasHandle_, HIPBLAS_TENSOR_OP_MATH);
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            spdlog::warn("Failed to enable tensor cores: {}", hipblasStatus_t);
        }
    }
    
    return true;
}

void CUDAKernelManager::shutdown() {
    // Destroy streams
    for (auto stream : streams_) {
        hipStreamDestroy(stream);
    }
    streams_.clear();
    
    // Destroy handles
    hipdnnDestroy(cudnnHandle_);
    hipblasDestroy(cublasHandle_);
}

bool CUDAKernelManager::setKernelConfig(const KernelConfig& config) {
    config_ = config;
    
    // Update tensor core settings
    if (config_.useTensorCores) {
        hipblasStatus_t status = hipblasSetMathMode(cublasHandle_, HIPBLAS_TENSOR_OP_MATH);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            spdlog::warn("Failed to enable tensor cores: {}", status);
            return false;
        }
    } else {
        hipblasStatus_t status = hipblasSetMathMode(cublasHandle_, HIPBLAS_DEFAULT_MATH);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            spdlog::warn("Failed to disable tensor cores: {}", status);
            return false;
        }
    }
    
    return true;
}

KernelConfig CUDAKernelManager::getKernelConfig() const {
    std::lock_guard<std::mutex> lock(mutex_);
    return config_;
}

bool CUDAKernelManager::enableTensorCores() {
    hipblasStatus_t status = hipblasSetMathMode(cublasHandle_, HIPBLAS_TENSOR_OP_MATH);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to enable tensor cores: {}", status);
        return false;
    }
    return true;
}

bool CUDAKernelManager::disableTensorCores() {
    hipblasStatus_t status = hipblasSetMathMode(cublasHandle_, HIPBLAS_DEFAULT_MATH);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to disable tensor cores: {}", status);
        return false;
    }
    return true;
}

bool CUDAKernelManager::areTensorCoresEnabled() const {
    return config_.useTensorCores;
}

bool CUDAKernelManager::enableMixedPrecision() {
    if (!checkMixedPrecisionSupport()) {
        spdlog::error("Mixed precision not supported on this device");
        return false;
    }

    config_.useMixedPrecision = true;
    return true;
}

void CUDAKernelManager::disableMixedPrecision() {
    config_.useMixedPrecision = false;
}

bool CUDAKernelManager::isMixedPrecisionEnabled() const {
    return config_.useMixedPrecision;
}

bool CUDAKernelManager::matrixMultiply(
    const void* A, const void* B, void* C,
    int m, int n, int k,
    hipDataType dataType
) {
    std::lock_guard<std::mutex> lock(mutex_);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;

    hipblasStatus_t status;
    if (dataType == HIP_R_16F && config_.useMixedPrecision) {
        status = hipblasHgemm(
            cublasHandle_, transA, transB,
            m, n, k,
            reinterpret_cast<const __half*>(&alpha),
            reinterpret_cast<const __half*>(A), m,
            reinterpret_cast<const __half*>(B), k,
            reinterpret_cast<const __half*>(&beta),
            reinterpret_cast<__half*>(C), m
        );
    } else {
        status = hipblasSgemm(
            cublasHandle_, transA, transB,
            m, n, k,
            &alpha,
            reinterpret_cast<const float*>(A), m,
            reinterpret_cast<const float*>(B), k,
            &beta,
            reinterpret_cast<float*>(C), m
        );
    }

    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Matrix multiplication failed: {}", cublasGetErrorString(status));
        return false;
    }

    return true;
}

bool CUDAKernelManager::convolutionForward(
    const void* input, const void* filter, void* output,
    int batchSize, int inChannels, int outChannels,
    int height, int width, int kernelSize,
    int stride, int padding,
    hipDataType dataType
) {
    std::lock_guard<std::mutex> lock(mutex_);

    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnConvolutionFwdAlgo_t algo;
    size_t workspaceSize = 0;
    void* workspace = nullptr;

    // Create descriptors
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnCreateConvolutionDescriptor(&convDesc);

    // Set tensor descriptors
    hipdnnSetTensor4dDescriptor(
        inputDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        batchSize,
        inChannels,
        height,
        width
    );

    hipdnnSetFilter4dDescriptor(
        filterDesc,
        getCudnnDataType(dataType),
        HIPDNN_TENSOR_NCHW,
        outChannels,
        inChannels,
        kernelSize,
        kernelSize
    );

    hipdnnSetConvolution2dDescriptor(
        convDesc,
        padding, padding,
        stride, stride,
        1, 1,
        HIPDNN_CROSS_CORRELATION,
        getCudnnDataType(dataType)
    );

    // Get output dimensions
    int outHeight, outWidth;
    hipdnnGetConvolution2dForwardOutputDim(
        convDesc,
        inputDesc,
        filterDesc,
        &batchSize,
        &outChannels,
        &outHeight,
        &outWidth
    );

    hipdnnSetTensor4dDescriptor(
        outputDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        batchSize,
        outChannels,
        outHeight,
        outWidth
    );

    // Find best algorithm
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    int returnedAlgoCount;
    hipdnnFindConvolutionForwardAlgorithm(
        cudnnHandle_,
        inputDesc,
        filterDesc,
        convDesc,
        outputDesc,
        1,
        &returnedAlgoCount,
        &perfResults
    );
    algo = perfResults.algo;

    // Get workspace size
    hipdnnGetConvolutionForwardWorkspaceSize(
        cudnnHandle_,
        inputDesc,
        filterDesc,
        convDesc,
        outputDesc,
        algo,
        &workspaceSize
    );

    if (workspaceSize > 0) {
        hipMalloc(&workspace, workspaceSize);
    }

    // Perform convolution
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipdnnStatus_t status = hipdnnConvolutionForward(
        cudnnHandle_,
        &alpha,
        inputDesc, input,
        filterDesc, filter,
        convDesc,
        algo,
        workspace,
        workspaceSize,
        &beta,
        outputDesc, output
    );

    // Cleanup
    if (workspace) {
        hipFree(workspace);
    }
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);

    if (status != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Convolution forward failed: {}", hipdnnGetErrorString(status));
        return false;
    }

    return true;
}

bool CUDAKernelManager::applyActivation(
    void* data, int size,
    const std::string& activationType,
    hipDataType dataType
) {
    std::lock_guard<std::mutex> lock(mutex_);

    hipdnnActivationDescriptor_t activationDesc;
    hipdnnTensorDescriptor_t dataDesc;

    hipdnnCreateActivationDescriptor(&activationDesc);
    hipdnnCreateTensorDescriptor(&dataDesc);

    hipdnnActivationMode_t mode;
    if (activationType == "relu") {
        mode = HIPDNN_ACTIVATION_RELU;
    } else if (activationType == "sigmoid") {
        mode = HIPDNN_ACTIVATION_SIGMOID;
    } else if (activationType == "tanh") {
        mode = HIPDNN_ACTIVATION_TANH;
    } else {
        spdlog::error("Unsupported activation type: {}", activationType);
        return false;
    }

    hipdnnSetActivationDescriptor(
        activationDesc,
        mode,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0
    );

    hipdnnSetTensor4dDescriptor(
        dataDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        1, 1, 1, size
    );

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipdnnStatus_t status = hipdnnActivationForward(
        cudnnHandle_,
        activationDesc,
        &alpha,
        dataDesc, data,
        &beta,
        dataDesc, data
    );

    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyTensorDescriptor(dataDesc);

    if (status != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Activation forward failed: {}", hipdnnGetErrorString(status));
        return false;
    }

    return true;
}

bool CUDAKernelManager::batchNormalization(
    void* data, const void* gamma, const void* beta,
    void* runningMean, void* runningVar,
    int batchSize, int channels, int spatialSize,
    float momentum, float epsilon,
    hipDataType dataType
) {
    std::lock_guard<std::mutex> lock(mutex_);

    hipdnnTensorDescriptor_t dataDesc, bnDesc;
    hipdnnCreateTensorDescriptor(&dataDesc);
    hipdnnCreateTensorDescriptor(&bnDesc);

    hipdnnSetTensor4dDescriptor(
        dataDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        batchSize,
        channels,
        spatialSize,
        spatialSize
    );

    hipdnnSetTensor4dDescriptor(
        bnDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        1,
        channels,
        1,
        1
    );

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipdnnStatus_t status = hipdnnBatchNormalizationForwardTraining(
        cudnnHandle_,
        HIPDNN_BATCHNORM_SPATIAL,
        &alpha,
        &beta,
        dataDesc, data,
        dataDesc, data,
        bnDesc,
        gamma,
        beta,
        momentum,
        runningMean,
        runningVar,
        epsilon,
        nullptr,
        nullptr
    );

    hipdnnDestroyTensorDescriptor(dataDesc);
    hipdnnDestroyTensorDescriptor(bnDesc);

    if (status != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Batch normalization failed: {}", hipdnnGetErrorString(status));
        return false;
    }

    return true;
}

bool CUDAKernelManager::selfAttention(
    const void* query, const void* key, const void* value,
    void* output,
    int batchSize, int seqLen, int headSize, int numHeads,
    hipDataType dataType
) {
    std::lock_guard<std::mutex> lock(mutex_);

    // Compute attention scores: Q * K^T
    void* attentionScores;
    hipMalloc(&attentionScores, batchSize * numHeads * seqLen * seqLen * sizeof(float));

    if (!matrixMultiply(
        query, key,
        attentionScores,
        batchSize * numHeads * seqLen,
        seqLen,
        headSize,
        dataType
    )) {
        hipFree(attentionScores);
        return false;
    }

    // Scale attention scores
    const float scale = 1.0f / std::sqrt(static_cast<float>(headSize));
    hipblasSscal(
        cublasHandle_,
        batchSize * numHeads * seqLen * seqLen,
        &scale,
        reinterpret_cast<float*>(attentionScores),
        1
    );

    // Apply softmax to attention scores
    if (!applyActivation(
        attentionScores,
        batchSize * numHeads * seqLen * seqLen,
        "softmax",
        dataType
    )) {
        hipFree(attentionScores);
        return false;
    }

    // Compute weighted sum: attention_scores * V
    if (!matrixMultiply(
        attentionScores,
        value,
        output,
        batchSize * numHeads * seqLen,
        headSize,
        seqLen,
        dataType
    )) {
        hipFree(attentionScores);
        return false;
    }

    hipFree(attentionScores);
    return true;
}

bool CUDAKernelManager::checkTensorCoreSupport() const {
    int deviceId;
    hipGetDevice(&deviceId);
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);
    
    // Check if device supports tensor cores (compute capability >= 7.0)
    return (prop.major > 7 || (prop.major == 7 && prop.minor >= 0));
}

bool CUDAKernelManager::checkMixedPrecisionSupport() const {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return prop.major >= 7;  // Volta architecture or newer
}

hipblasComputeType_t CUDAKernelManager::getComputeType() const {
    if (config_.useTensorCores && config_.useMixedPrecision) {
        return HIPBLAS_COMPUTE_32F_FAST_TF32;
    } else if (config_.useTensorCores) {
        return HIPBLAS_COMPUTE_32F;
    } else {
        return HIPBLAS_COMPUTE_32F;
    }
}

hipdnnDataType_t CUDAKernelManager::getCudnnDataType(hipDataType dataType) const {
    switch (dataType) {
        case HIP_R_16F:
            return HIPDNN_DATA_HALF;
        case HIP_R_32F:
            return HIPDNN_DATA_FLOAT;
        case HIP_R_64F:
            return HIPDNN_DATA_DOUBLE;
        default:
            return HIPDNN_DATA_FLOAT;
    }
}

} // namespace msmartcompute 