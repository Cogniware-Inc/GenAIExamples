#include "hip/hip_runtime.h"
#include "cuda_loss.h"
#include "cuda_kernels.cu"
#include <spdlog/spdlog.h>

namespace msmartcompute {

// CUDA Cross Entropy Loss Implementation
CUDACrossEntropyLoss::CUDACrossEntropyLoss() {
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDACrossEntropyLoss::~CUDACrossEntropyLoss() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

float CUDACrossEntropyLoss::compute(const std::vector<float>& outputs, const std::vector<float>& targets) {
    // Resize device vectors if needed
    if (deviceOutputs_.size() != outputs.size()) {
        deviceOutputs_.resize(outputs.size());
        deviceTargets_.resize(targets.size());
        deviceLoss_.resize(outputs.size());
    }

    // Copy data to device
    thrust::copy(outputs.begin(), outputs.end(), deviceOutputs_.begin());
    thrust::copy(targets.begin(), targets.end(), deviceTargets_.begin());

    // Compute softmax
    thrust::device_vector<float> expOutputs(outputs.size());
    thrust::transform(deviceOutputs_.begin(), deviceOutputs_.end(), expOutputs.begin(),
        [](float x) { return std::exp(x); });
    
    float sum = thrust::reduce(expOutputs.begin(), expOutputs.end(), 0.0f, thrust::plus<float>());
    
    thrust::transform(expOutputs.begin(), expOutputs.end(), deviceOutputs_.begin(),
        [sum](float x) { return x / sum; });

    // Compute cross entropy loss
    thrust::transform(deviceOutputs_.begin(), deviceOutputs_.end(), deviceTargets_.begin(),
        deviceLoss_.begin(), [](float pred, float target) {
            return -target * std::log(pred + 1e-8f);
        });

    // Reduce loss
    float totalLoss = thrust::reduce(deviceLoss_.begin(), deviceLoss_.end(), 0.0f, thrust::plus<float>());
    return totalLoss / outputs.size();
}

// CUDA MSE Loss Implementation
CUDAMSELoss::CUDAMSELoss() {
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDAMSELoss::~CUDAMSELoss() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

float CUDAMSELoss::compute(const std::vector<float>& outputs, const std::vector<float>& targets) {
    // Resize device vectors if needed
    if (deviceOutputs_.size() != outputs.size()) {
        deviceOutputs_.resize(outputs.size());
        deviceTargets_.resize(targets.size());
        deviceLoss_.resize(outputs.size());
    }

    // Copy data to device
    thrust::copy(outputs.begin(), outputs.end(), deviceOutputs_.begin());
    thrust::copy(targets.begin(), targets.end(), deviceTargets_.begin());

    // Compute squared differences
    thrust::transform(deviceOutputs_.begin(), deviceOutputs_.end(), deviceTargets_.begin(),
        deviceLoss_.begin(), [](float pred, float target) {
            float diff = pred - target;
            return diff * diff;
        });

    // Reduce loss
    float totalLoss = thrust::reduce(deviceLoss_.begin(), deviceLoss_.end(), 0.0f, thrust::plus<float>());
    return totalLoss / outputs.size();
}

// CUDA Binary Cross Entropy Loss Implementation
CUDABinaryCrossEntropyLoss::CUDABinaryCrossEntropyLoss() {
    CUDA_CHECK(hipStreamCreate(&stream_));
}

CUDABinaryCrossEntropyLoss::~CUDABinaryCrossEntropyLoss() {
    CUDA_CHECK(hipStreamDestroy(stream_));
}

float CUDABinaryCrossEntropyLoss::compute(const std::vector<float>& outputs, const std::vector<float>& targets) {
    // Resize device vectors if needed
    if (deviceOutputs_.size() != outputs.size()) {
        deviceOutputs_.resize(outputs.size());
        deviceTargets_.resize(targets.size());
        deviceLoss_.resize(outputs.size());
    }

    // Copy data to device
    thrust::copy(outputs.begin(), outputs.end(), deviceOutputs_.begin());
    thrust::copy(targets.begin(), targets.end(), deviceTargets_.begin());

    // Apply sigmoid to outputs
    thrust::transform(deviceOutputs_.begin(), deviceOutputs_.end(), deviceOutputs_.begin(),
        [](float x) { return 1.0f / (1.0f + std::exp(-x)); });

    // Compute binary cross entropy loss
    thrust::transform(deviceOutputs_.begin(), deviceOutputs_.end(), deviceTargets_.begin(),
        deviceLoss_.begin(), [](float pred, float target) {
            return -(target * std::log(pred + 1e-8f) + 
                    (1.0f - target) * std::log(1.0f - pred + 1e-8f));
        });

    // Reduce loss
    float totalLoss = thrust::reduce(deviceLoss_.begin(), deviceLoss_.end(), 0.0f, thrust::plus<float>());
    return totalLoss / outputs.size();
}

} // namespace msmartcompute 