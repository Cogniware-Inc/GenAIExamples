#include "memory_virtualization_manager.h"
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <thread>
#include <mutex>
#include <queue>
#include <unordered_map>
#include <memory>

namespace msmartcompute {

MemoryVirtualizationManager& MemoryVirtualizationManager::getInstance() {
    static MemoryVirtualizationManager instance;
    return instance;
}

bool MemoryVirtualizationManager::initialize(const MemoryVirtualizationConfig& config) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    config_ = config;
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to set CUDA device: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceProperties(&prop, config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to get device properties: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    deviceProps_ = prop;
    
    // Initialize memory pools
    if (!initializeMemoryPools()) {
        spdlog::error("Failed to initialize memory pools");
        return false;
    }
    
    // Initialize page tables
    if (!initializePageTables()) {
        spdlog::error("Failed to initialize page tables");
        return false;
    }
    
    // Initialize memory defragmentation
    if (!initializeDefragmentation()) {
        spdlog::error("Failed to initialize defragmentation");
        return false;
    }
    
    // Start memory monitoring thread
    running_ = true;
    monitoringThread_ = std::thread(&MemoryVirtualizationManager::monitoringLoop, this);
    
    spdlog::info("Memory Virtualization Manager initialized successfully");
    return true;
}

void MemoryVirtualizationManager::shutdown() {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (!running_) return;
    
    running_ = false;
    
    // Stop monitoring thread
    if (monitoringThread_.joinable()) {
        monitoringThread_.join();
    }
    
    // Cleanup memory pools
    cleanupMemoryPools();
    
    // Cleanup page tables
    cleanupPageTables();
    
    // Cleanup defragmentation
    cleanupDefragmentation();
    
    spdlog::info("Memory Virtualization Manager shutdown completed");
}

bool MemoryVirtualizationManager::createVirtualMemorySpace(int virtualGPUId, size_t size) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    // Check if virtual memory space already exists
    if (virtualMemorySpaces_.find(virtualGPUId) != virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} already exists", virtualGPUId);
        return false;
    }
    
    // Create virtual memory space
    VirtualMemorySpace space;
    space.virtualGPUId = virtualGPUId;
    space.totalSize = size;
    space.allocatedSize = 0;
    space.freeSize = size;
    space.pageTable = std::make_unique<PageTable>();
    
    // Initialize page table
    if (!space.pageTable->initialize(size, config_.pageSize)) {
        spdlog::error("Failed to initialize page table for virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Allocate physical memory pool
    hipError_t status = hipMalloc(&space.physicalMemoryPool, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to allocate physical memory pool for virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    virtualMemorySpaces_[virtualGPUId] = space;
    
    spdlog::info("Virtual memory space created for GPU {} with size {} MB", 
                 virtualGPUId, size / (1024 * 1024));
    return true;
}

bool MemoryVirtualizationManager::destroyVirtualMemorySpace(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Free all allocated memory
    for (auto& allocation : space.allocations) {
        if (allocation.physicalAddress) {
            hipFree(allocation.physicalAddress);
        }
    }
    space.allocations.clear();
    
    // Free physical memory pool
    if (space.physicalMemoryPool) {
        hipFree(space.physicalMemoryPool);
        space.physicalMemoryPool = nullptr;
    }
    
    // Destroy page table
    space.pageTable->shutdown();
    
    virtualMemorySpaces_.erase(it);
    
    spdlog::info("Virtual memory space destroyed for GPU {}", virtualGPUId);
    return true;
}

void* MemoryVirtualizationManager::allocateMemory(int virtualGPUId, size_t size, size_t alignment) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return nullptr;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Check if enough memory is available
    if (space.allocatedSize + size > space.totalSize) {
        spdlog::error("Insufficient memory in virtual GPU {}: requested {} bytes, available {} bytes", 
                     virtualGPUId, size, space.freeSize);
        return nullptr;
    }
    
    // Allocate physical memory
    void* physicalAddress = nullptr;
    hipError_t status = hipMalloc(&physicalAddress, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to allocate physical memory for virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return nullptr;
    }
    
    // Create virtual address mapping
    void* virtualAddress = space.pageTable->allocateVirtualAddress(size, alignment);
    if (!virtualAddress) {
        spdlog::error("Failed to allocate virtual address for virtual GPU {}", virtualGPUId);
        hipFree(physicalAddress);
        return nullptr;
    }
    
    // Map virtual address to physical address
    if (!space.pageTable->mapVirtualToPhysical(virtualAddress, physicalAddress, size)) {
        spdlog::error("Failed to map virtual to physical address for virtual GPU {}", virtualGPUId);
        hipFree(physicalAddress);
        space.pageTable->freeVirtualAddress(virtualAddress);
        return nullptr;
    }
    
    // Track allocation
    MemoryAllocation alloc;
    alloc.virtualAddress = virtualAddress;
    alloc.physicalAddress = physicalAddress;
    alloc.size = size;
    alloc.alignment = alignment;
    alloc.timestamp = std::chrono::steady_clock::now();
    space.allocations.push_back(alloc);
    
    // Update memory statistics
    space.allocatedSize += size;
    space.freeSize = space.totalSize - space.allocatedSize;
    
    spdlog::debug("Allocated {} bytes in virtual GPU {}: virtual={}, physical={}", 
                  size, virtualGPUId, virtualAddress, physicalAddress);
    
    return virtualAddress;
}

bool MemoryVirtualizationManager::freeMemory(int virtualGPUId, void* virtualAddress) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Find allocation
    auto allocIt = std::find_if(space.allocations.begin(), space.allocations.end(),
        [virtualAddress](const MemoryAllocation& alloc) {
            return alloc.virtualAddress == virtualAddress;
        });
    
    if (allocIt == space.allocations.end()) {
        spdlog::error("Memory allocation not found in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Free physical memory
    hipError_t status = hipFree(allocIt->physicalAddress);
    if (status != hipSuccess) {
        spdlog::error("Failed to free physical memory for virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    // Unmap virtual address
    if (!space.pageTable->unmapVirtualAddress(virtualAddress)) {
        spdlog::error("Failed to unmap virtual address for virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Free virtual address
    space.pageTable->freeVirtualAddress(virtualAddress);
    
    // Update memory statistics
    space.allocatedSize -= allocIt->size;
    space.freeSize = space.totalSize - space.allocatedSize;
    
    // Remove allocation
    space.allocations.erase(allocIt);
    
    spdlog::debug("Freed memory in virtual GPU {}: virtual={}", virtualGPUId, virtualAddress);
    return true;
}

bool MemoryVirtualizationManager::copyMemory(int virtualGPUId, 
                                            void* dst, const void* src, 
                                            size_t size, hipMemcpyKind kind) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Get physical addresses
    void* physicalDst = space.pageTable->getPhysicalAddress(dst);
    void* physicalSrc = space.pageTable->getPhysicalAddress(const_cast<void*>(src));
    
    if (!physicalDst || !physicalSrc) {
        spdlog::error("Failed to get physical addresses for memory copy in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Perform memory copy
    hipError_t status = hipMemcpy(physicalDst, physicalSrc, size, kind);
    if (status != hipSuccess) {
        spdlog::error("Failed to copy memory in virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    return true;
}

bool MemoryVirtualizationManager::memset(int virtualGPUId, void* virtualAddress, int value, size_t size) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Get physical address
    void* physicalAddress = space.pageTable->getPhysicalAddress(virtualAddress);
    if (!physicalAddress) {
        spdlog::error("Failed to get physical address for memset in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Perform memset
    hipError_t status = hipMemset(physicalAddress, value, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to memset in virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    return true;
}

VirtualMemoryInfo MemoryVirtualizationManager::getVirtualMemoryInfo(int virtualGPUId) const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    VirtualMemoryInfo info;
    info.virtualGPUId = virtualGPUId;
    info.totalSize = 0;
    info.allocatedSize = 0;
    info.freeSize = 0;
    info.fragmentationLevel = 0.0f;
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        return info;
    }
    
    const VirtualMemorySpace& space = it->second;
    info.totalSize = space.totalSize;
    info.allocatedSize = space.allocatedSize;
    info.freeSize = space.freeSize;
    info.fragmentationLevel = calculateFragmentationLevel(space);
    
    return info;
}

std::vector<VirtualMemoryInfo> MemoryVirtualizationManager::getAllVirtualMemoryInfo() const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    std::vector<VirtualMemoryInfo> infos;
    infos.reserve(virtualMemorySpaces_.size());
    
    for (const auto& pair : virtualMemorySpaces_) {
        infos.push_back(getVirtualMemoryInfo(pair.first));
    }
    
    return infos;
}

bool MemoryVirtualizationManager::defragment(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualMemorySpaces_.find(virtualGPUId);
    if (it == virtualMemorySpaces_.end()) {
        spdlog::error("Virtual memory space for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualMemorySpace& space = it->second;
    
    // Perform defragmentation
    if (!defragmentationEngine_->defragment(space)) {
        spdlog::error("Failed to defragment virtual memory space for GPU {}", virtualGPUId);
        return false;
    }
    
    spdlog::info("Defragmentation completed for virtual GPU {}", virtualGPUId);
    return true;
}

bool MemoryVirtualizationManager::initializeMemoryPools() {
    // Initialize different memory pools for different allocation sizes
    memoryPools_.resize(config_.numMemoryPools);
    
    for (int i = 0; i < config_.numMemoryPools; ++i) {
        size_t poolSize = config_.basePoolSize * (1 << i);  // Exponential growth
        size_t blockSize = config_.baseBlockSize * (1 << i);
        
        MemoryPool& pool = memoryPools_[i];
        pool.blockSize = blockSize;
        pool.totalSize = poolSize;
        pool.allocatedSize = 0;
        pool.freeSize = poolSize;
        
        // Allocate pool memory
        hipError_t status = hipMalloc(&pool.memory, poolSize);
        if (status != hipSuccess) {
            spdlog::error("Failed to allocate memory pool {}: {}", i, hipGetErrorString(status));
            return false;
        }
        
        // Initialize free blocks
        pool.freeBlocks.clear();
        size_t numBlocks = poolSize / blockSize;
        for (size_t j = 0; j < numBlocks; ++j) {
            void* blockAddress = static_cast<char*>(pool.memory) + (j * blockSize);
            pool.freeBlocks.push_back(blockAddress);
        }
    }
    
    spdlog::info("Memory pools initialized with {} pools", config_.numMemoryPools);
    return true;
}

bool MemoryVirtualizationManager::initializePageTables() {
    // Initialize page table management
    pageTableManager_ = std::make_unique<PageTableManager>();
    
    if (!pageTableManager_->initialize(config_.pageSize, config_.maxPages)) {
        spdlog::error("Failed to initialize page table manager");
        return false;
    }
    
    spdlog::info("Page tables initialized with page size {} bytes", config_.pageSize);
    return true;
}

bool MemoryVirtualizationManager::initializeDefragmentation() {
    // Initialize defragmentation engine
    defragmentationEngine_ = std::make_unique<DefragmentationEngine>();
    
    if (!defragmentationEngine_->initialize(config_.defragmentationThreshold)) {
        spdlog::error("Failed to initialize defragmentation engine");
        return false;
    }
    
    spdlog::info("Defragmentation engine initialized with threshold {}", config_.defragmentationThreshold);
    return true;
}

void MemoryVirtualizationManager::cleanupMemoryPools() {
    for (auto& pool : memoryPools_) {
        if (pool.memory) {
            hipFree(pool.memory);
            pool.memory = nullptr;
        }
        pool.freeBlocks.clear();
    }
    memoryPools_.clear();
}

void MemoryVirtualizationManager::cleanupPageTables() {
    if (pageTableManager_) {
        pageTableManager_->shutdown();
        pageTableManager_.reset();
    }
}

void MemoryVirtualizationManager::cleanupDefragmentation() {
    if (defragmentationEngine_) {
        defragmentationEngine_->shutdown();
        defragmentationEngine_.reset();
    }
}

void MemoryVirtualizationManager::monitoringLoop() {
    while (running_) {
        // Update memory statistics
        updateMemoryStatistics();
        
        // Check for fragmentation
        checkFragmentation();
        
        // Perform automatic defragmentation if needed
        performAutomaticDefragmentation();
        
        // Sleep for monitoring interval
        std::this_thread::sleep_for(std::chrono::milliseconds(config_.monitoringInterval));
    }
}

void MemoryVirtualizationManager::updateMemoryStatistics() {
    for (auto& pair : virtualMemorySpaces_) {
        VirtualMemorySpace& space = pair.second;
        
        // Update allocation statistics
        space.allocatedSize = 0;
        for (const auto& alloc : space.allocations) {
            space.allocatedSize += alloc.size;
        }
        space.freeSize = space.totalSize - space.allocatedSize;
    }
}

void MemoryVirtualizationManager::checkFragmentation() {
    for (auto& pair : virtualMemorySpaces_) {
        int virtualGPUId = pair.first;
        VirtualMemorySpace& space = pair.second;
        
        float fragmentationLevel = calculateFragmentationLevel(space);
        if (fragmentationLevel > config_.defragmentationThreshold) {
            spdlog::warn("High fragmentation detected in virtual GPU {}: {:.2f}%", 
                        virtualGPUId, fragmentationLevel * 100.0f);
        }
    }
}

void MemoryVirtualizationManager::performAutomaticDefragmentation() {
    for (auto& pair : virtualMemorySpaces_) {
        int virtualGPUId = pair.first;
        VirtualMemorySpace& space = pair.second;
        
        float fragmentationLevel = calculateFragmentationLevel(space);
        if (fragmentationLevel > config_.defragmentationThreshold && 
            config_.enableAutomaticDefragmentation) {
            
            spdlog::info("Performing automatic defragmentation for virtual GPU {}", virtualGPUId);
            defragment(virtualGPUId);
        }
    }
}

float MemoryVirtualizationManager::calculateFragmentationLevel(const VirtualMemorySpace& space) const {
    if (space.allocations.empty()) {
        return 0.0f;
    }
    
    // Calculate fragmentation based on allocation patterns
    size_t totalGaps = 0;
    size_t totalAllocated = 0;
    
    // Sort allocations by address
    std::vector<MemoryAllocation> sortedAllocs = space.allocations;
    std::sort(sortedAllocs.begin(), sortedAllocs.end(),
        [](const MemoryAllocation& a, const MemoryAllocation& b) {
            return a.virtualAddress < b.virtualAddress;
        });
    
    // Calculate gaps between allocations
    for (size_t i = 0; i < sortedAllocs.size() - 1; ++i) {
        uintptr_t currentEnd = reinterpret_cast<uintptr_t>(sortedAllocs[i].virtualAddress) + 
                              sortedAllocs[i].size;
        uintptr_t nextStart = reinterpret_cast<uintptr_t>(sortedAllocs[i + 1].virtualAddress);
        
        if (nextStart > currentEnd) {
            totalGaps += nextStart - currentEnd;
        }
        totalAllocated += sortedAllocs[i].size;
    }
    
    if (totalAllocated == 0) {
        return 0.0f;
    }
    
    return static_cast<float>(totalGaps) / totalAllocated;
}

} // namespace msmartcompute 