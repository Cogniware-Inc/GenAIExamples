#include "hip/hip_runtime.h"
#include "cuda_virtualization_driver.h"
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <nvml.h>
#include <algorithm>
#include <chrono>
#include <thread>
#include <mutex>
#include <queue>
#include <unordered_map>
#include <memory>

namespace msmartcompute {

// CUDA Virtualization Driver Implementation
CUDAVirtualizationDriver& CUDAVirtualizationDriver::getInstance() {
    static CUDAVirtualizationDriver instance;
    return instance;
}

bool CUDAVirtualizationDriver::initialize(const VirtualizationConfig& config) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    config_ = config;
    
    // Initialize NVML for GPU monitoring
    nvmlReturn_t nvmlStatus = nvmlInit();
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to initialize NVML: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    // Get number of GPUs
    unsigned int deviceCount;
    nvmlStatus = nvmlDeviceGetCount(&deviceCount);
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to get device count: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    if (config_.deviceId >= deviceCount) {
        spdlog::error("Invalid device ID: {} (max: {})", config_.deviceId, deviceCount - 1);
        return false;
    }
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to set CUDA device: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceProperties(&prop, config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to get device properties: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    deviceProps_ = prop;
    
    // Initialize NVML device handle
    nvmlStatus = nvmlDeviceGetHandleByIndex(config_.deviceId, &nvmlDevice_);
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to get NVML device handle: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    // Initialize cuBLAS
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle_);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuBLAS handle: {}", hipblasStatus_t);
        return false;
    }
    
    // Initialize cuDNN
    hipdnnStatus_t cudnnStatus = hipdnnCreate(&cudnnHandle_);
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuDNN handle: {}", hipdnnGetErrorString(cudnnStatus));
        return false;
    }
    
    // Create CUDA streams for virtualization
    streams_.resize(config_.numVirtualStreams);
    for (int i = 0; i < config_.numVirtualStreams; ++i) {
        cudaStatus = hipStreamCreate(&streams_[i]);
        if (cudaStatus != hipSuccess) {
            spdlog::error("Failed to create CUDA stream {}: {}", i, hipGetErrorString(cudaStatus));
            return false;
        }
    }
    
    // Initialize virtual GPU contexts
    if (!initializeVirtualGPUContexts()) {
        spdlog::error("Failed to initialize virtual GPU contexts");
        return false;
    }
    
    // Initialize memory virtualization
    if (!initializeMemoryVirtualization()) {
        spdlog::error("Failed to initialize memory virtualization");
        return false;
    }
    
    // Initialize compute virtualization
    if (!initializeComputeVirtualization()) {
        spdlog::error("Failed to initialize compute virtualization");
        return false;
    }
    
    // Start monitoring thread
    running_ = true;
    monitoringThread_ = std::thread(&CUDAVirtualizationDriver::monitoringLoop, this);
    
    spdlog::info("CUDA Virtualization Driver initialized successfully");
    return true;
}

void CUDAVirtualizationDriver::shutdown() {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (!running_) return;
    
    running_ = false;
    
    // Stop monitoring thread
    if (monitoringThread_.joinable()) {
        monitoringThread_.join();
    }
    
    // Cleanup virtual GPU contexts
    cleanupVirtualGPUContexts();
    
    // Cleanup memory virtualization
    cleanupMemoryVirtualization();
    
    // Cleanup compute virtualization
    cleanupComputeVirtualization();
    
    // Destroy CUDA streams
    for (auto stream : streams_) {
        hipStreamDestroy(stream);
    }
    streams_.clear();
    
    // Destroy handles
    hipdnnDestroy(cudnnHandle_);
    hipblasDestroy(cublasHandle_);
    
    // Shutdown NVML
    nvmlShutdown();
    
    spdlog::info("CUDA Virtualization Driver shutdown completed");
}

bool CUDAVirtualizationDriver::createVirtualGPU(const VirtualGPUConfig& config) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    // Check if virtual GPU ID already exists
    if (virtualGPUs_.find(config.virtualGPUId) != virtualGPUs_.end()) {
        spdlog::error("Virtual GPU with ID {} already exists", config.virtualGPUId);
        return false;
    }
    
    // Create virtual GPU context
    VirtualGPUContext context;
    context.config = config;
    context.status = VirtualGPUStatus::CREATED;
    context.memoryAllocated = 0;
    context.computeUtilization = 0.0f;
    context.memoryUtilization = 0.0f;
    context.activeStreams = 0;
    
    // Allocate memory for virtual GPU
    if (config.memoryLimit > 0) {
        hipError_t status = hipMalloc(&context.memoryPool, config.memoryLimit);
        if (status != hipSuccess) {
            spdlog::error("Failed to allocate memory for virtual GPU {}: {}", 
                         config.virtualGPUId, hipGetErrorString(status));
            return false;
        }
        context.memoryLimit = config.memoryLimit;
    }
    
    // Create CUDA streams for virtual GPU
    context.streams.resize(config.numStreams);
    for (int i = 0; i < config.numStreams; ++i) {
        hipError_t status = hipStreamCreate(&context.streams[i]);
        if (status != hipSuccess) {
            spdlog::error("Failed to create stream for virtual GPU {}: {}", 
                         config.virtualGPUId, hipGetErrorString(status));
            return false;
        }
    }
    
    // Create cuBLAS handle for virtual GPU
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&context.cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuBLAS handle for virtual GPU {}: {}", 
                     config.virtualGPUId, hipblasStatus_t);
        return false;
    }
    
    // Create cuDNN handle for virtual GPU
    hipdnnStatus_t cudnnStatus = hipdnnCreate(&context.cudnnHandle);
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuDNN handle for virtual GPU {}: {}", 
                     config.virtualGPUId, hipdnnGetErrorString(cudnnStatus));
        return false;
    }
    
    // Set tensor core mode if enabled
    if (config.enableTensorCores) {
        hipblasStatus_t = hipblasSetMathMode(context.cublasHandle, HIPBLAS_TENSOR_OP_MATH);
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            spdlog::warn("Failed to enable tensor cores for virtual GPU {}: {}", 
                        config.virtualGPUId, hipblasStatus_t);
        }
    }
    
    virtualGPUs_[config.virtualGPUId] = context;
    
    spdlog::info("Virtual GPU {} created successfully", config.virtualGPUId);
    return true;
}

bool CUDAVirtualizationDriver::destroyVirtualGPU(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        spdlog::error("Virtual GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualGPUContext& context = it->second;
    
    // Destroy cuDNN handle
    hipdnnDestroy(context.cudnnHandle);
    
    // Destroy cuBLAS handle
    hipblasDestroy(context.cublasHandle);
    
    // Destroy CUDA streams
    for (auto stream : context.streams) {
        hipStreamDestroy(stream);
    }
    context.streams.clear();
    
    // Free memory pool
    if (context.memoryPool) {
        hipFree(context.memoryPool);
        context.memoryPool = nullptr;
    }
    
    virtualGPUs_.erase(it);
    
    spdlog::info("Virtual GPU {} destroyed successfully", virtualGPUId);
    return true;
}

bool CUDAVirtualizationDriver::allocateMemory(int virtualGPUId, size_t size, void** ptr) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        spdlog::error("Virtual GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualGPUContext& context = it->second;
    
    // Check memory limit
    if (context.memoryAllocated + size > context.memoryLimit) {
        spdlog::error("Memory allocation failed: insufficient memory in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Allocate memory
    hipError_t status = hipMalloc(ptr, size);
    if (status != hipSuccess) {
        spdlog::error("Failed to allocate memory in virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    // Update memory tracking
    context.memoryAllocated += size;
    context.memoryUtilization = static_cast<float>(context.memoryAllocated) / context.memoryLimit;
    
    // Track allocation
    MemoryAllocation alloc;
    alloc.ptr = *ptr;
    alloc.size = size;
    alloc.timestamp = std::chrono::steady_clock::now();
    context.memoryAllocations.push_back(alloc);
    
    return true;
}

bool CUDAVirtualizationDriver::freeMemory(int virtualGPUId, void* ptr) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        spdlog::error("Virtual GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualGPUContext& context = it->second;
    
    // Find and remove allocation
    auto allocIt = std::find_if(context.memoryAllocations.begin(), 
                               context.memoryAllocations.end(),
                               [ptr](const MemoryAllocation& alloc) {
                                   return alloc.ptr == ptr;
                               });
    
    if (allocIt == context.memoryAllocations.end()) {
        spdlog::error("Memory allocation not found in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Free memory
    hipError_t status = hipFree(ptr);
    if (status != hipSuccess) {
        spdlog::error("Failed to free memory in virtual GPU {}: {}", 
                     virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    // Update memory tracking
    context.memoryAllocated -= allocIt->size;
    context.memoryUtilization = static_cast<float>(context.memoryAllocated) / context.memoryLimit;
    context.memoryAllocations.erase(allocIt);
    
    return true;
}

bool CUDAVirtualizationDriver::matrixMultiply(int virtualGPUId,
                                             const void* A, const void* B, void* C,
                                             int m, int n, int k,
                                             hipDataType dataType,
                                             int streamId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        spdlog::error("Virtual GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualGPUContext& context = it->second;
    
    if (streamId >= context.streams.size()) {
        spdlog::error("Invalid stream ID {} for virtual GPU {}", streamId, virtualGPUId);
        return false;
    }
    
    // Set stream for cuBLAS
    hipblasSetStream(context.cublasHandle, context.streams[streamId]);
    
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    
    hipblasStatus_t status;
    if (dataType == HIP_R_16F && context.config.enableMixedPrecision) {
        status = hipblasHgemm(
            context.cublasHandle, transA, transB,
            m, n, k,
            reinterpret_cast<const __half*>(&alpha),
            reinterpret_cast<const __half*>(A), m,
            reinterpret_cast<const __half*>(B), k,
            reinterpret_cast<const __half*>(&beta),
            reinterpret_cast<__half*>(C), m
        );
    } else {
        status = hipblasSgemm(
            context.cublasHandle, transA, transB,
            m, n, k,
            &alpha,
            reinterpret_cast<const float*>(A), m,
            reinterpret_cast<const float*>(B), k,
            &beta,
            reinterpret_cast<float*>(C), m
        );
    }
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Matrix multiplication failed in virtual GPU {}: {}", 
                     virtualGPUId, cublasGetErrorString(status));
        return false;
    }
    
    // Update compute utilization
    context.computeUtilization = std::min(1.0f, context.computeUtilization + 0.1f);
    
    return true;
}

bool CUDAVirtualizationDriver::convolutionForward(int virtualGPUId,
                                                 const void* input, const void* filter, void* output,
                                                 int batchSize, int inChannels, int outChannels,
                                                 int height, int width, int kernelSize,
                                                 int stride, int padding,
                                                 hipDataType dataType,
                                                 int streamId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        spdlog::error("Virtual GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualGPUContext& context = it->second;
    
    if (streamId >= context.streams.size()) {
        spdlog::error("Invalid stream ID {} for virtual GPU {}", streamId, virtualGPUId);
        return false;
    }
    
    // Set stream for cuDNN
    hipdnnSetStream(context.cudnnHandle, context.streams[streamId]);
    
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnConvolutionFwdAlgo_t algo;
    size_t workspaceSize = 0;
    void* workspace = nullptr;
    
    // Create descriptors
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnCreateConvolutionDescriptor(&convDesc);
    
    // Set tensor descriptors
    hipdnnSetTensor4dDescriptor(
        inputDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        batchSize,
        inChannels,
        height,
        width
    );
    
    hipdnnSetFilter4dDescriptor(
        filterDesc,
        getCudnnDataType(dataType),
        HIPDNN_TENSOR_NCHW,
        outChannels,
        inChannels,
        kernelSize,
        kernelSize
    );
    
    hipdnnSetConvolution2dDescriptor(
        convDesc,
        padding, padding,
        stride, stride,
        1, 1,
        HIPDNN_CROSS_CORRELATION,
        getCudnnDataType(dataType)
    );
    
    // Get output dimensions
    int outHeight, outWidth;
    hipdnnGetConvolution2dForwardOutputDim(
        convDesc,
        inputDesc,
        filterDesc,
        &batchSize,
        &outChannels,
        &outHeight,
        &outWidth
    );
    
    hipdnnSetTensor4dDescriptor(
        outputDesc,
        HIPDNN_TENSOR_NCHW,
        getCudnnDataType(dataType),
        batchSize,
        outChannels,
        outHeight,
        outWidth
    );
    
    // Find best algorithm
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    int returnedAlgoCount;
    hipdnnFindConvolutionForwardAlgorithm(
        context.cudnnHandle,
        inputDesc,
        filterDesc,
        convDesc,
        outputDesc,
        1,
        &returnedAlgoCount,
        &perfResults
    );
    algo = perfResults.algo;
    
    // Get workspace size
    hipdnnGetConvolutionForwardWorkspaceSize(
        context.cudnnHandle,
        inputDesc,
        filterDesc,
        convDesc,
        outputDesc,
        algo,
        &workspaceSize
    );
    
    if (workspaceSize > 0) {
        hipMalloc(&workspace, workspaceSize);
    }
    
    // Perform convolution
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    hipdnnStatus_t status = hipdnnConvolutionForward(
        context.cudnnHandle,
        &alpha,
        inputDesc, input,
        filterDesc, filter,
        convDesc,
        algo,
        workspace,
        workspaceSize,
        &beta,
        outputDesc, output
    );
    
    // Cleanup
    if (workspace) {
        hipFree(workspace);
    }
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    
    if (status != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Convolution forward failed in virtual GPU {}: {}", 
                     virtualGPUId, hipdnnGetErrorString(status));
        return false;
    }
    
    // Update compute utilization
    context.computeUtilization = std::min(1.0f, context.computeUtilization + 0.15f);
    
    return true;
}

VirtualGPUStatus CUDAVirtualizationDriver::getVirtualGPUStatus(int virtualGPUId) const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        return VirtualGPUStatus::NOT_FOUND;
    }
    
    return it->second.status;
}

VirtualGPUInfo CUDAVirtualizationDriver::getVirtualGPUInfo(int virtualGPUId) const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    VirtualGPUInfo info;
    info.virtualGPUId = virtualGPUId;
    info.status = VirtualGPUStatus::NOT_FOUND;
    
    auto it = virtualGPUs_.find(virtualGPUId);
    if (it == virtualGPUs_.end()) {
        return info;
    }
    
    const VirtualGPUContext& context = it->second;
    info.status = context.status;
    info.memoryAllocated = context.memoryAllocated;
    info.memoryLimit = context.memoryLimit;
    info.memoryUtilization = context.memoryUtilization;
    info.computeUtilization = context.computeUtilization;
    info.activeStreams = context.activeStreams;
    info.numStreams = context.streams.size();
    
    return info;
}

std::vector<VirtualGPUInfo> CUDAVirtualizationDriver::getAllVirtualGPUInfo() const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    std::vector<VirtualGPUInfo> infos;
    infos.reserve(virtualGPUs_.size());
    
    for (const auto& pair : virtualGPUs_) {
        infos.push_back(getVirtualGPUInfo(pair.first));
    }
    
    return infos;
}

bool CUDAVirtualizationDriver::initializeVirtualGPUContexts() {
    // Initialize virtual GPU context management
    virtualGPUContexts_.reserve(config_.maxVirtualGPUs);
    
    spdlog::info("Virtual GPU contexts initialized");
    return true;
}

bool CUDAVirtualizationDriver::initializeMemoryVirtualization() {
    // Initialize memory virtualization system
    memoryVirtualizationEnabled_ = true;
    
    // Get total GPU memory
    size_t totalMemory;
    hipError_t status = hipMemGetInfo(&freeMemory_, &totalMemory);
    if (status != hipSuccess) {
        spdlog::error("Failed to get GPU memory info: {}", hipGetErrorString(status));
        return false;
    }
    
    totalMemory_ = totalMemory;
    usedMemory_ = totalMemory - freeMemory_;
    
    spdlog::info("Memory virtualization initialized - Total: {} MB, Used: {} MB", 
                 totalMemory_ / (1024 * 1024), usedMemory_ / (1024 * 1024));
    return true;
}

bool CUDAVirtualizationDriver::initializeComputeVirtualization() {
    // Initialize compute virtualization system
    computeVirtualizationEnabled_ = true;
    
    // Get GPU compute capability
    spdlog::info("Compute virtualization initialized - Compute Capability: {}.{}", 
                 deviceProps_.major, deviceProps_.minor);
    return true;
}

void CUDAVirtualizationDriver::cleanupVirtualGPUContexts() {
    // Cleanup all virtual GPU contexts
    for (auto& pair : virtualGPUs_) {
        VirtualGPUContext& context = pair.second;
        
        // Destroy cuDNN handle
        hipdnnDestroy(context.cudnnHandle);
        
        // Destroy cuBLAS handle
        hipblasDestroy(context.cublasHandle);
        
        // Destroy CUDA streams
        for (auto stream : context.streams) {
            hipStreamDestroy(stream);
        }
        
        // Free memory pool
        if (context.memoryPool) {
            hipFree(context.memoryPool);
        }
    }
    
    virtualGPUs_.clear();
    virtualGPUContexts_.clear();
}

void CUDAVirtualizationDriver::cleanupMemoryVirtualization() {
    memoryVirtualizationEnabled_ = false;
    totalMemory_ = 0;
    usedMemory_ = 0;
    freeMemory_ = 0;
}

void CUDAVirtualizationDriver::cleanupComputeVirtualization() {
    computeVirtualizationEnabled_ = false;
}

void CUDAVirtualizationDriver::monitoringLoop() {
    while (running_) {
        // Update GPU utilization
        updateGPUUtilization();
        
        // Update memory usage
        updateMemoryUsage();
        
        // Update virtual GPU status
        updateVirtualGPUStatus();
        
        // Sleep for monitoring interval
        std::this_thread::sleep_for(std::chrono::milliseconds(config_.monitoringInterval));
    }
}

void CUDAVirtualizationDriver::updateGPUUtilization() {
    unsigned int utilization;
    nvmlReturn_t status = nvmlDeviceGetUtilizationRates(nvmlDevice_, &utilization);
    if (status == NVML_SUCCESS) {
        gpuUtilization_ = static_cast<float>(utilization) / 100.0f;
    }
}

void CUDAVirtualizationDriver::updateMemoryUsage() {
    size_t free, total;
    hipError_t status = hipMemGetInfo(&free, &total);
    if (status == hipSuccess) {
        freeMemory_ = free;
        totalMemory_ = total;
        usedMemory_ = total - free;
    }
}

void CUDAVirtualizationDriver::updateVirtualGPUStatus() {
    for (auto& pair : virtualGPUs_) {
        VirtualGPUContext& context = pair.second;
        
        // Update compute utilization (decay over time)
        context.computeUtilization = std::max(0.0f, context.computeUtilization - 0.01f);
        
        // Update active streams count
        context.activeStreams = 0;
        for (auto stream : context.streams) {
            hipError_t status = hipStreamQuery(stream);
            if (status == hipErrorNotReady) {
                context.activeStreams++;
            }
        }
    }
}

hipdnnDataType_t CUDAVirtualizationDriver::getCudnnDataType(hipDataType dataType) const {
    switch (dataType) {
        case HIP_R_16F:
            return HIPDNN_DATA_HALF;
        case HIP_R_32F:
            return HIPDNN_DATA_FLOAT;
        case HIP_R_64F:
            return HIPDNN_DATA_DOUBLE;
        default:
            return HIPDNN_DATA_FLOAT;
    }
}

} // namespace msmartcompute 