#include "hip/hip_runtime.h"
#include "compute_virtualization_manager.h"
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <algorithm>
#include <chrono>
#include <thread>
#include <mutex>
#include <queue>
#include <unordered_map>
#include <memory>

namespace msmartcompute {

ComputeVirtualizationManager& ComputeVirtualizationManager::getInstance() {
    static ComputeVirtualizationManager instance;
    return instance;
}

bool ComputeVirtualizationManager::initialize(const ComputeVirtualizationConfig& config) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    config_ = config;
    
    // Initialize NVML for GPU monitoring
    nvmlReturn_t nvmlStatus = nvmlInit();
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to initialize NVML: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    // Get number of GPUs
    unsigned int deviceCount;
    nvmlStatus = nvmlDeviceGetCount(&deviceCount);
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to get device count: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    if (config_.deviceId >= deviceCount) {
        spdlog::error("Invalid device ID: {} (max: {})", config_.deviceId, deviceCount - 1);
        return false;
    }
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to set CUDA device: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceProperties(&prop, config_.deviceId);
    if (cudaStatus != hipSuccess) {
        spdlog::error("Failed to get device properties: {}", hipGetErrorString(cudaStatus));
        return false;
    }
    
    deviceProps_ = prop;
    
    // Initialize NVML device handle
    nvmlStatus = nvmlDeviceGetHandleByIndex(config_.deviceId, &nvmlDevice_);
    if (nvmlStatus != NVML_SUCCESS) {
        spdlog::error("Failed to get NVML device handle: {}", nvmlErrorString(nvmlStatus));
        return false;
    }
    
    // Initialize compute units
    if (!initializeComputeUnits()) {
        spdlog::error("Failed to initialize compute units");
        return false;
    }
    
    // Initialize scheduler
    if (!initializeScheduler()) {
        spdlog::error("Failed to initialize scheduler");
        return false;
    }
    
    // Initialize load balancer
    if (!initializeLoadBalancer()) {
        spdlog::error("Failed to initialize load balancer");
        return false;
    }
    
    // Start monitoring thread
    running_ = true;
    monitoringThread_ = std::thread(&ComputeVirtualizationManager::monitoringLoop, this);
    
    spdlog::info("Compute Virtualization Manager initialized successfully");
    return true;
}

void ComputeVirtualizationManager::shutdown() {
    std::lock_guard<std::mutex> lock(mutex_);
    
    if (!running_) return;
    
    running_ = false;
    
    // Stop monitoring thread
    if (monitoringThread_.joinable()) {
        monitoringThread_.join();
    }
    
    // Cleanup compute units
    cleanupComputeUnits();
    
    // Cleanup scheduler
    cleanupScheduler();
    
    // Cleanup load balancer
    cleanupLoadBalancer();
    
    // Shutdown NVML
    nvmlShutdown();
    
    spdlog::info("Compute Virtualization Manager shutdown completed");
}

bool ComputeVirtualizationManager::createVirtualComputeUnit(int virtualGPUId, const VirtualComputeUnitConfig& config) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    // Check if virtual compute unit already exists
    if (virtualComputeUnits_.find(virtualGPUId) != virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} already exists", virtualGPUId);
        return false;
    }
    
    // Create virtual compute unit
    VirtualComputeUnit unit;
    unit.virtualGPUId = virtualGPUId;
    unit.config = config;
    unit.status = VirtualComputeUnitStatus::CREATED;
    unit.computeUtilization = 0.0f;
    unit.memoryUtilization = 0.0f;
    unit.activeKernels = 0;
    unit.totalKernelsExecuted = 0;
    
    // Allocate compute resources
    if (!allocateComputeResources(unit)) {
        spdlog::error("Failed to allocate compute resources for virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Create CUDA streams
    unit.streams.resize(config.numStreams);
    for (int i = 0; i < config.numStreams; ++i) {
        hipError_t status = hipStreamCreate(&unit.streams[i]);
        if (status != hipSuccess) {
            spdlog::error("Failed to create stream for virtual GPU {}: {}", 
                         virtualGPUId, hipGetErrorString(status));
            return false;
        }
    }
    
    // Create cuBLAS handle
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&unit.cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuBLAS handle for virtual GPU {}: {}", 
                     virtualGPUId, hipblasStatus_t);
        return false;
    }
    
    // Create cuDNN handle
    hipdnnStatus_t cudnnStatus = hipdnnCreate(&unit.cudnnHandle);
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        spdlog::error("Failed to create cuDNN handle for virtual GPU {}: {}", 
                     virtualGPUId, hipdnnGetErrorString(cudnnStatus));
        return false;
    }
    
    // Set tensor core mode if enabled
    if (config.enableTensorCores) {
        hipblasStatus_t = hipblasSetMathMode(unit.cublasHandle, HIPBLAS_TENSOR_OP_MATH);
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
            spdlog::warn("Failed to enable tensor cores for virtual GPU {}: {}", 
                        virtualGPUId, hipblasStatus_t);
        }
    }
    
    virtualComputeUnits_[virtualGPUId] = unit;
    
    spdlog::info("Virtual compute unit created for GPU {} with {} compute units", 
                 virtualGPUId, config.numComputeUnits);
    return true;
}

bool ComputeVirtualizationManager::destroyVirtualComputeUnit(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    // Destroy cuDNN handle
    hipdnnDestroy(unit.cudnnHandle);
    
    // Destroy cuBLAS handle
    hipblasDestroy(unit.cublasHandle);
    
    // Destroy CUDA streams
    for (auto stream : unit.streams) {
        hipStreamDestroy(stream);
    }
    unit.streams.clear();
    
    // Free compute resources
    freeComputeResources(unit);
    
    virtualComputeUnits_.erase(it);
    
    spdlog::info("Virtual compute unit destroyed for GPU {}", virtualGPUId);
    return true;
}

bool ComputeVirtualizationManager::executeKernel(int virtualGPUId, 
                                                const KernelConfig& kernelConfig,
                                                int streamId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    if (streamId >= unit.streams.size()) {
        spdlog::error("Invalid stream ID {} for virtual GPU {}", streamId, virtualGPUId);
        return false;
    }
    
    // Check compute resource availability
    if (!checkComputeResourceAvailability(unit, kernelConfig)) {
        spdlog::error("Insufficient compute resources for kernel execution in virtual GPU {}", virtualGPUId);
        return false;
    }
    
    // Schedule kernel execution
    KernelExecution execution;
    execution.kernelConfig = kernelConfig;
    execution.streamId = streamId;
    execution.startTime = std::chrono::steady_clock::now();
    execution.status = KernelExecutionStatus::QUEUED;
    
    // Add to execution queue
    unit.kernelQueue.push(execution);
    
    // Update statistics
    unit.activeKernels++;
    unit.totalKernelsExecuted++;
    
    // Update compute utilization
    unit.computeUtilization = std::min(1.0f, unit.computeUtilization + 0.05f);
    
    spdlog::debug("Kernel queued for execution in virtual GPU {} on stream {}", virtualGPUId, streamId);
    return true;
}

bool ComputeVirtualizationManager::synchronize(int virtualGPUId, int streamId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    if (streamId >= unit.streams.size()) {
        spdlog::error("Invalid stream ID {} for virtual GPU {}", streamId, virtualGPUId);
        return false;
    }
    
    // Synchronize stream
    hipError_t status = hipStreamSynchronize(unit.streams[streamId]);
    if (status != hipSuccess) {
        spdlog::error("Failed to synchronize stream {} in virtual GPU {}: {}", 
                     streamId, virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    // Update kernel execution status
    updateKernelExecutionStatus(unit, streamId);
    
    return true;
}

VirtualComputeUnitInfo ComputeVirtualizationManager::getVirtualComputeUnitInfo(int virtualGPUId) const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    VirtualComputeUnitInfo info;
    info.virtualGPUId = virtualGPUId;
    info.status = VirtualComputeUnitStatus::NOT_FOUND;
    info.computeUtilization = 0.0f;
    info.memoryUtilization = 0.0f;
    info.activeKernels = 0;
    info.totalKernelsExecuted = 0;
    info.numStreams = 0;
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        return info;
    }
    
    const VirtualComputeUnit& unit = it->second;
    info.status = unit.status;
    info.computeUtilization = unit.computeUtilization;
    info.memoryUtilization = unit.memoryUtilization;
    info.activeKernels = unit.activeKernels;
    info.totalKernelsExecuted = unit.totalKernelsExecuted;
    info.numStreams = unit.streams.size();
    info.numComputeUnits = unit.config.numComputeUnits;
    
    return info;
}

std::vector<VirtualComputeUnitInfo> ComputeVirtualizationManager::getAllVirtualComputeUnitInfo() const {
    std::lock_guard<std::mutex> lock(mutex_);
    
    std::vector<VirtualComputeUnitInfo> infos;
    infos.reserve(virtualComputeUnits_.size());
    
    for (const auto& pair : virtualComputeUnits_) {
        infos.push_back(getVirtualComputeUnitInfo(pair.first));
    }
    
    return infos;
}

bool ComputeVirtualizationManager::setComputeShare(int virtualGPUId, float computeShare) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    if (computeShare < 0.0f || computeShare > 1.0f) {
        spdlog::error("Invalid compute share: {} (must be between 0.0 and 1.0)", computeShare);
        return false;
    }
    
    unit.config.computeShare = computeShare;
    
    // Update scheduler
    scheduler_->updateComputeShare(virtualGPUId, computeShare);
    
    spdlog::info("Compute share updated for virtual GPU {}: {:.2f}%", virtualGPUId, computeShare * 100.0f);
    return true;
}

bool ComputeVirtualizationManager::enableTensorCores(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    hipblasStatus_t status = hipblasSetMathMode(unit.cublasHandle, HIPBLAS_TENSOR_OP_MATH);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to enable tensor cores for virtual GPU {}: {}", virtualGPUId, status);
        return false;
    }
    
    unit.config.enableTensorCores = true;
    
    spdlog::info("Tensor cores enabled for virtual GPU {}", virtualGPUId);
    return true;
}

bool ComputeVirtualizationManager::disableTensorCores(int virtualGPUId) {
    std::lock_guard<std::mutex> lock(mutex_);
    
    auto it = virtualComputeUnits_.find(virtualGPUId);
    if (it == virtualComputeUnits_.end()) {
        spdlog::error("Virtual compute unit for GPU {} not found", virtualGPUId);
        return false;
    }
    
    VirtualComputeUnit& unit = it->second;
    
    hipblasStatus_t status = hipblasSetMathMode(unit.cublasHandle, HIPBLAS_DEFAULT_MATH);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        spdlog::error("Failed to disable tensor cores for virtual GPU {}: {}", virtualGPUId, status);
        return false;
    }
    
    unit.config.enableTensorCores = false;
    
    spdlog::info("Tensor cores disabled for virtual GPU {}", virtualGPUId);
    return true;
}

bool ComputeVirtualizationManager::initializeComputeUnits() {
    // Initialize compute unit management
    computeUnitManager_ = std::make_unique<ComputeUnitManager>();
    
    if (!computeUnitManager_->initialize(deviceProps_.multiProcessorCount, config_.maxVirtualComputeUnits)) {
        spdlog::error("Failed to initialize compute unit manager");
        return false;
    }
    
    spdlog::info("Compute units initialized with {} physical compute units", deviceProps_.multiProcessorCount);
    return true;
}

bool ComputeVirtualizationManager::initializeScheduler() {
    // Initialize compute scheduler
    scheduler_ = std::make_unique<ComputeScheduler>();
    
    if (!scheduler_->initialize(config_.schedulingPolicy, config_.timeSlice)) {
        spdlog::error("Failed to initialize compute scheduler");
        return false;
    }
    
    spdlog::info("Compute scheduler initialized with policy {}", config_.schedulingPolicy);
    return true;
}

bool ComputeVirtualizationManager::initializeLoadBalancer() {
    // Initialize load balancer
    loadBalancer_ = std::make_unique<LoadBalancer>();
    
    if (!loadBalancer_->initialize(config_.loadBalancingStrategy)) {
        spdlog::error("Failed to initialize load balancer");
        return false;
    }
    
    spdlog::info("Load balancer initialized with strategy {}", config_.loadBalancingStrategy);
    return true;
}

void ComputeVirtualizationManager::cleanupComputeUnits() {
    if (computeUnitManager_) {
        computeUnitManager_->shutdown();
        computeUnitManager_.reset();
    }
}

void ComputeVirtualizationManager::cleanupScheduler() {
    if (scheduler_) {
        scheduler_->shutdown();
        scheduler_.reset();
    }
}

void ComputeVirtualizationManager::cleanupLoadBalancer() {
    if (loadBalancer_) {
        loadBalancer_->shutdown();
        loadBalancer_.reset();
    }
}

void ComputeVirtualizationManager::monitoringLoop() {
    while (running_) {
        // Update GPU utilization
        updateGPUUtilization();
        
        // Update compute unit statistics
        updateComputeUnitStatistics();
        
        // Perform load balancing
        performLoadBalancing();
        
        // Process kernel queue
        processKernelQueue();
        
        // Sleep for monitoring interval
        std::this_thread::sleep_for(std::chrono::milliseconds(config_.monitoringInterval));
    }
}

void ComputeVirtualizationManager::updateGPUUtilization() {
    unsigned int utilization;
    nvmlReturn_t status = nvmlDeviceGetUtilizationRates(nvmlDevice_, &utilization);
    if (status == NVML_SUCCESS) {
        gpuUtilization_ = static_cast<float>(utilization) / 100.0f;
    }
}

void ComputeVirtualizationManager::updateComputeUnitStatistics() {
    for (auto& pair : virtualComputeUnits_) {
        VirtualComputeUnit& unit = pair.second;
        
        // Update compute utilization (decay over time)
        unit.computeUtilization = std::max(0.0f, unit.computeUtilization - 0.01f);
        
        // Update active kernels count
        unit.activeKernels = 0;
        for (auto stream : unit.streams) {
            hipError_t status = hipStreamQuery(stream);
            if (status == hipErrorNotReady) {
                unit.activeKernels++;
            }
        }
    }
}

void ComputeVirtualizationManager::performLoadBalancing() {
    if (!loadBalancer_) return;
    
    // Get current load distribution
    std::vector<LoadInfo> loadInfos;
    for (const auto& pair : virtualComputeUnits_) {
        const VirtualComputeUnit& unit = pair.second;
        LoadInfo info;
        info.virtualGPUId = unit.virtualGPUId;
        info.computeUtilization = unit.computeUtilization;
        info.activeKernels = unit.activeKernels;
        loadInfos.push_back(info);
    }
    
    // Perform load balancing
    auto balancingActions = loadBalancer_->balance(loadInfos);
    
    // Apply balancing actions
    for (const auto& action : balancingActions) {
        applyLoadBalancingAction(action);
    }
}

void ComputeVirtualizationManager::processKernelQueue() {
    for (auto& pair : virtualComputeUnits_) {
        VirtualComputeUnit& unit = pair.second;
        
        while (!unit.kernelQueue.empty()) {
            KernelExecution& execution = unit.kernelQueue.front();
            
            // Check if kernel can be executed
            if (canExecuteKernel(unit, execution)) {
                // Execute kernel
                if (executeKernelOnDevice(unit, execution)) {
                    execution.status = KernelExecutionStatus::RUNNING;
                    execution.startTime = std::chrono::steady_clock::now();
                } else {
                    execution.status = KernelExecutionStatus::FAILED;
                }
                
                unit.kernelQueue.pop();
            } else {
                // Kernel cannot be executed yet, keep in queue
                break;
            }
        }
    }
}

bool ComputeVirtualizationManager::allocateComputeResources(VirtualComputeUnit& unit) {
    // Allocate compute units from the manager
    if (!computeUnitManager_->allocateComputeUnits(unit.virtualGPUId, unit.config.numComputeUnits)) {
        spdlog::error("Failed to allocate compute units for virtual GPU {}", unit.virtualGPUId);
        return false;
    }
    
    return true;
}

void ComputeVirtualizationManager::freeComputeResources(VirtualComputeUnit& unit) {
    // Free compute units
    computeUnitManager_->freeComputeUnits(unit.virtualGPUId);
}

bool ComputeVirtualizationManager::checkComputeResourceAvailability(const VirtualComputeUnit& unit, 
                                                                   const KernelConfig& kernelConfig) {
    // Check if enough compute resources are available
    return unit.activeKernels < unit.config.maxConcurrentKernels;
}

void ComputeVirtualizationManager::updateKernelExecutionStatus(VirtualComputeUnit& unit, int streamId) {
    // Update kernel execution status based on stream completion
    for (auto& execution : unit.kernelExecutions) {
        if (execution.streamId == streamId && execution.status == KernelExecutionStatus::RUNNING) {
            execution.status = KernelExecutionStatus::COMPLETED;
            execution.endTime = std::chrono::steady_clock::now();
            unit.activeKernels--;
        }
    }
}

bool ComputeVirtualizationManager::canExecuteKernel(const VirtualComputeUnit& unit, 
                                                   const KernelExecution& execution) {
    // Check if kernel can be executed based on resource availability
    return unit.activeKernels < unit.config.maxConcurrentKernels;
}

bool ComputeVirtualizationManager::executeKernelOnDevice(VirtualComputeUnit& unit, 
                                                        KernelExecution& execution) {
    // Execute kernel on the device
    // This is a simplified implementation - in practice, you would launch actual CUDA kernels
    hipError_t status = hipSuccess; // Placeholder for actual kernel execution
    
    if (status != hipSuccess) {
        spdlog::error("Failed to execute kernel on virtual GPU {}: {}", 
                     unit.virtualGPUId, hipGetErrorString(status));
        return false;
    }
    
    return true;
}

void ComputeVirtualizationManager::applyLoadBalancingAction(const LoadBalancingAction& action) {
    // Apply load balancing action
    switch (action.type) {
        case LoadBalancingActionType::MIGRATE_KERNEL:
            migrateKernel(action.sourceGPUId, action.targetGPUId, action.kernelId);
            break;
        case LoadBalancingActionType::ADJUST_COMPUTE_SHARE:
            setComputeShare(action.targetGPUId, action.computeShare);
            break;
        default:
            spdlog::warn("Unknown load balancing action type");
            break;
    }
}

void ComputeVirtualizationManager::migrateKernel(int sourceGPUId, int targetGPUId, int kernelId) {
    // Migrate kernel from source to target GPU
    spdlog::info("Migrating kernel {} from virtual GPU {} to {}", kernelId, sourceGPUId, targetGPUId);
    // Implementation would involve moving kernel execution context
}

} // namespace msmartcompute 