#include "hip/hip_runtime.h"
#include "llm_inference_core/transformer/transformer_block.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>

namespace msmartcompute {

namespace cg = cooperative_groups;

// Helper functions for CUDA kernels
__device__ float gelu(float x) {
    return 0.5f * x * (1.0f + tanhf(sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
}

__device__ float silu(float x) {
    return x / (1.0f + expf(-x));
}

// Layer normalization kernel
__global__ void layerNormKernel(const float* input,
                              float* output,
                              const float* scale,
                              const float* bias,
                              int hiddenSize,
                              float epsilon = 1e-5f) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    extern __shared__ float sdata[];
    float* mean = sdata;
    float* var = &sdata[1];
    
    // Compute mean
    float sum = 0.0f;
    for (int i = tid; i < hiddenSize; i += blockDim.x) {
        sum += input[bid * hiddenSize + i];
    }
    sum = blockReduceSum(sum);
    
    if (tid == 0) {
        *mean = sum / hiddenSize;
    }
    __syncthreads();
    
    // Compute variance
    float sqSum = 0.0f;
    for (int i = tid; i < hiddenSize; i += blockDim.x) {
        float diff = input[bid * hiddenSize + i] - *mean;
        sqSum += diff * diff;
    }
    sqSum = blockReduceSum(sqSum);
    
    if (tid == 0) {
        *var = sqSum / hiddenSize + epsilon;
    }
    __syncthreads();
    
    // Normalize and scale
    for (int i = tid; i < hiddenSize; i += blockDim.x) {
        float normalized = (input[bid * hiddenSize + i] - *mean) / sqrtf(*var);
        output[bid * hiddenSize + i] = scale[i] * normalized + bias[i];
    }
}

// Self-attention kernel
__global__ void attentionKernel(const float* input,
                              float* output,
                              const float* weights,
                              float* kv_cache,
                              int batchSize,
                              int seqLength,
                              int hiddenSize,
                              int numHeads,
                              float scale) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    // Compute Q, K, V projections
    float* q = input + bid * hiddenSize;
    float* k = input + bid * hiddenSize + hiddenSize;
    float* v = input + bid * hiddenSize + 2 * hiddenSize;
    
    // Compute attention scores
    float* scores = output + bid * numHeads * seqLength;
    for (int h = 0; h < numHeads; ++h) {
        for (int i = tid; i < seqLength; i += blockDim.x) {
            float score = 0.0f;
            for (int j = 0; j < hiddenSize / numHeads; ++j) {
                score += q[h * (hiddenSize / numHeads) + j] * k[i * (hiddenSize / numHeads) + j];
            }
            scores[h * seqLength + i] = score * scale;
        }
    }
    __syncthreads();
    
    // Apply softmax
    for (int h = 0; h < numHeads; ++h) {
        float maxScore = -INFINITY;
        for (int i = tid; i < seqLength; i += blockDim.x) {
            maxScore = max(maxScore, scores[h * seqLength + i]);
        }
        maxScore = blockReduceMax(maxScore);
        
        float sum = 0.0f;
        for (int i = tid; i < seqLength; i += blockDim.x) {
            scores[h * seqLength + i] = expf(scores[h * seqLength + i] - maxScore);
            sum += scores[h * seqLength + i];
        }
        sum = blockReduceSum(sum);
        
        for (int i = tid; i < seqLength; i += blockDim.x) {
            scores[h * seqLength + i] /= sum;
        }
    }
    __syncthreads();
    
    // Compute output
    float* out = output + bid * hiddenSize;
    for (int h = 0; h < numHeads; ++h) {
        for (int i = tid; i < hiddenSize / numHeads; i += blockDim.x) {
            float sum = 0.0f;
            for (int j = 0; j < seqLength; ++j) {
                sum += scores[h * seqLength + j] * v[j * (hiddenSize / numHeads) + i];
            }
            out[h * (hiddenSize / numHeads) + i] = sum;
        }
    }
}

// Feed-forward network kernel
__global__ void ffnKernel(const float* input,
                         float* output,
                         const float* weights1,
                         const float* weights2,
                         int hiddenSize,
                         int intermediateSize,
                         const char* activationType) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    // First layer
    float* intermediate = output + bid * intermediateSize;
    for (int i = tid; i < intermediateSize; i += blockDim.x) {
        float sum = 0.0f;
        for (int j = 0; j < hiddenSize; ++j) {
            sum += input[bid * hiddenSize + j] * weights1[j * intermediateSize + i];
        }
        
        // Apply activation
        if (strcmp(activationType, "gelu") == 0) {
            intermediate[i] = gelu(sum);
        } else if (strcmp(activationType, "silu") == 0) {
            intermediate[i] = silu(sum);
        } else {
            intermediate[i] = sum;  // Linear activation
        }
    }
    __syncthreads();
    
    // Second layer
    float* out = output + bid * hiddenSize;
    for (int i = tid; i < hiddenSize; i += blockDim.x) {
        float sum = 0.0f;
        for (int j = 0; j < intermediateSize; ++j) {
            sum += intermediate[j] * weights2[j * hiddenSize + i];
        }
        out[i] = sum;
    }
}

// Helper function for block reduction
template<typename T>
__device__ T blockReduceSum(T val) {
    cg::thread_block block = cg::this_thread_block();
    T sum = val;
    
    for (int offset = block.size() / 2; offset > 0; offset /= 2) {
        sum += block.shfl_down(sum, offset);
    }
    
    return sum;
}

template<typename T>
__device__ T blockReduceMax(T val) {
    cg::thread_block block = cg::this_thread_block();
    T max_val = val;
    
    for (int offset = block.size() / 2; offset > 0; offset /= 2) {
        max_val = max(max_val, block.shfl_down(max_val, offset));
    }
    
    return max_val;
}

bool TransformerBlock::launchLayerNormKernel(const float* input,
                                           float* output,
                                           int batchSize,
                                           int seqLength,
                                           hipStream_t stream) {
    int blockSize = 256;
    int numBlocks = batchSize * seqLength;
    int sharedMemSize = 2 * sizeof(float);  // For mean and variance
    
    layerNormKernel<<<numBlocks, blockSize, sharedMemSize, stream>>>(
        input, output,
        d_weights_,  // Scale
        d_weights_ + config_.hiddenSize,  // Bias
        config_.hiddenSize
    );
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        lastError_ = hipGetErrorString(error);
        return false;
    }
    
    return true;
}

bool TransformerBlock::launchAttentionKernel(const float* input,
                                           float* output,
                                           int batchSize,
                                           int seqLength,
                                           hipStream_t stream) {
    int blockSize = 256;
    int numBlocks = batchSize;
    float scale = 1.0f / sqrtf(config_.hiddenSize / config_.numHeads);
    
    attentionKernel<<<numBlocks, blockSize, 0, stream>>>(
        input, output,
        d_weights_,
        d_kv_cache_,
        batchSize,
        seqLength,
        config_.hiddenSize,
        config_.numHeads,
        scale
    );
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        lastError_ = hipGetErrorString(error);
        return false;
    }
    
    return true;
}

bool TransformerBlock::launchFFNKernel(const float* input,
                                     float* output,
                                     int batchSize,
                                     int seqLength,
                                     hipStream_t stream) {
    int blockSize = 256;
    int numBlocks = batchSize;
    
    ffnKernel<<<numBlocks, blockSize, 0, stream>>>(
        input, output,
        d_weights_,  // First layer weights
        d_weights_ + config_.hiddenSize * config_.intermediateSize,  // Second layer weights
        config_.hiddenSize,
        config_.intermediateSize,
        config_.activationType.c_str()
    );
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        lastError_ = hipGetErrorString(error);
        return false;
    }
    
    return true;
}

} // namespace msmartcompute 