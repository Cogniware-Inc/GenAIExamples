#include "hip/hip_runtime.h"
#include "activation_kernels.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>

namespace msmartcompute {
namespace llm_inference {

// CUDA kernel for ReLU activation
__global__ void reluKernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = max(0.0f, input[idx]);
    }
}

__global__ void reluKernel(half* output, const half* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __hmax(__float2half(0.0f), input[idx]);
    }
}

// CUDA kernel for GELU activation
__global__ void geluKernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanhf(sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

__global__ void geluKernel(half* output, const half* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = __half2float(input[idx]);
        output[idx] = __float2half(0.5f * x * (1.0f + tanhf(sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x))));
    }
}

// CUDA kernel for SiLU (Swish) activation
__global__ void siluKernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = x / (1.0f + expf(-x));
    }
}

__global__ void siluKernel(half* output, const half* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = __half2float(input[idx]);
        output[idx] = __float2half(x / (1.0f + expf(-x)));
    }
}

// CUDA kernel for Tanh activation
__global__ void tanhKernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = tanhf(input[idx]);
    }
}

__global__ void tanhKernel(half* output, const half* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(tanhf(__half2float(input[idx])));
    }
}

// CUDA kernel for Sigmoid activation
__global__ void sigmoidKernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));
    }
}

__global__ void sigmoidKernel(half* output, const half* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = __half2float(input[idx]);
        output[idx] = __float2half(1.0f / (1.0f + expf(-x)));
    }
}

// CUDA kernel for Softmax activation
__global__ void softmaxKernel(float* output, const float* input, int batch_size, int seq_len, int hidden_size) {
    int batch_idx = blockIdx.x;
    int seq_idx = blockIdx.y;
    int tid = threadIdx.x;
    
    extern __shared__ float sdata[];
    
    // Load input data
    if (tid < hidden_size) {
        sdata[tid] = input[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid];
    }
    __syncthreads();
    
    // Find maximum value
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset && tid + offset < hidden_size) {
            sdata[tid] = max(sdata[tid], sdata[tid + offset]);
        }
        __syncthreads();
    }
    
    float max_val = sdata[0];
    __syncthreads();
    
    // Compute exp and sum
    if (tid < hidden_size) {
        sdata[tid] = expf(input[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid] - max_val);
    }
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset && tid + offset < hidden_size) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }
    
    float sum = sdata[0];
    __syncthreads();
    
    // Normalize
    if (tid < hidden_size) {
        output[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid] = sdata[tid] / sum;
    }
}

__global__ void softmaxKernel(half* output, const half* input, int batch_size, int seq_len, int hidden_size) {
    int batch_idx = blockIdx.x;
    int seq_idx = blockIdx.y;
    int tid = threadIdx.x;
    
    extern __shared__ float sdata[];
    
    // Load input data
    if (tid < hidden_size) {
        sdata[tid] = __half2float(input[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid]);
    }
    __syncthreads();
    
    // Find maximum value
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset && tid + offset < hidden_size) {
            sdata[tid] = max(sdata[tid], sdata[tid + offset]);
        }
        __syncthreads();
    }
    
    float max_val = sdata[0];
    __syncthreads();
    
    // Compute exp and sum
    if (tid < hidden_size) {
        sdata[tid] = expf(__half2float(input[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid]) - max_val);
    }
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset && tid + offset < hidden_size) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }
    
    float sum = sdata[0];
    __syncthreads();
    
    // Normalize
    if (tid < hidden_size) {
        output[batch_idx * seq_len * hidden_size + seq_idx * hidden_size + tid] = __float2half(sdata[tid] / sum);
    }
}

// Kernel launcher implementations
void launchReLU(float* output, const float* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    reluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchReLU(half* output, const half* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    reluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchGELU(float* output, const float* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    geluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchGELU(half* output, const half* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    geluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchSiLU(float* output, const float* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    siluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchSiLU(half* output, const half* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    siluKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchTanh(float* output, const float* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    tanhKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchTanh(half* output, const half* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    tanhKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchSigmoid(float* output, const float* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    sigmoidKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchSigmoid(half* output, const half* input, int size, hipStream_t stream) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    sigmoidKernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void launchSoftmax(float* output, const float* input, int batch_size, int seq_len, int hidden_size, hipStream_t stream) {
    dim3 block(256);
    dim3 grid(batch_size, seq_len);
    size_t shared_mem_size = hidden_size * sizeof(float);
    
    softmaxKernel<<<grid, block, shared_mem_size, stream>>>(output, input, batch_size, seq_len, hidden_size);
    CUDA_CHECK(hipGetLastError());
}

void launchSoftmax(half* output, const half* input, int batch_size, int seq_len, int hidden_size, hipStream_t stream) {
    dim3 block(256);
    dim3 grid(batch_size, seq_len);
    size_t shared_mem_size = hidden_size * sizeof(float);
    
    softmaxKernel<<<grid, block, shared_mem_size, stream>>>(output, input, batch_size, seq_len, hidden_size);
    CUDA_CHECK(hipGetLastError());
}

// Generic activation launcher
void launchActivation(float* output, const float* input, int size, ActivationType type, hipStream_t stream) {
    switch (type) {
        case ActivationType::RELU:
            launchReLU(output, input, size, stream);
            break;
        case ActivationType::GELU:
            launchGELU(output, input, size, stream);
            break;
        case ActivationType::SILU:
            launchSiLU(output, input, size, stream);
            break;
        case ActivationType::TANH:
            launchTanh(output, input, size, stream);
            break;
        case ActivationType::SIGMOID:
            launchSigmoid(output, input, size, stream);
            break;
        default:
            throw std::runtime_error("Unsupported activation type");
    }
}

void launchActivation(half* output, const half* input, int size, ActivationType type, hipStream_t stream) {
    switch (type) {
        case ActivationType::RELU:
            launchReLU(output, input, size, stream);
            break;
        case ActivationType::GELU:
            launchGELU(output, input, size, stream);
            break;
        case ActivationType::SILU:
            launchSiLU(output, input, size, stream);
            break;
        case ActivationType::TANH:
            launchTanh(output, input, size, stream);
            break;
        case ActivationType::SIGMOID:
            launchSigmoid(output, input, size, stream);
            break;
        default:
            throw std::runtime_error("Unsupported activation type");
    }
}

} // namespace llm_inference
} // namespace msmartcompute
