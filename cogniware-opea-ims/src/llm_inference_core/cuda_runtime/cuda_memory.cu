#include "hip/hip_runtime.h"
#include "cuda_memory.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>
#include <unordered_map>
#include <mutex>
#include <stdexcept>

namespace msmartcompute {
namespace llm_inference {

// Implementation of CUDAMemoryManager
struct CUDAMemoryManager::Impl {
    std::unordered_map<void*, MemoryAllocation> allocations;
    std::unordered_map<int, size_t> device_memory_limits;
    std::unordered_map<int, size_t> device_memory_used;
    std::mutex mutex;
};

CUDAMemoryManager& CUDAMemoryManager::getInstance() {
    static CUDAMemoryManager instance;
    return instance;
}

CUDAMemoryManager::CUDAMemoryManager() : pimpl(std::make_unique<Impl>()) {
    // Initialize memory manager
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

CUDAMemoryManager::~CUDAMemoryManager() {
    clear();
}

void CUDAMemoryManager::initializeDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Set device
    setDevice(device_id);
    
    // Get total memory
    size_t free, total;
    getDeviceMemoryInfo(free, total, device_id);
    
    // Set default memory limit to 90% of total memory
    pimpl->device_memory_limits[device_id] = total * 0.9;
    pimpl->device_memory_used[device_id] = 0;
    
    spdlog::info("Initialized CUDA memory manager for device {}: {} MB total, {} MB limit",
                 device_id, total / (1024 * 1024), pimpl->device_memory_limits[device_id] / (1024 * 1024));
}

void CUDAMemoryManager::cleanupDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Free all allocations for this device
    for (auto it = pimpl->allocations.begin(); it != pimpl->allocations.end();) {
        if (it->second.device_id == device_id) {
            deallocate(it->first);
            it = pimpl->allocations.erase(it);
        } else {
            ++it;
        }
    }
    
    // Reset memory stats
    pimpl->device_memory_used[device_id] = 0;
}

void CUDAMemoryManager::checkMemoryLimit(size_t size, int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (pimpl->device_memory_used[device_id] + size > pimpl->device_memory_limits[device_id]) {
        throw std::runtime_error("Memory allocation would exceed device limit");
    }
}

void CUDAMemoryManager::updateMemoryStats(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    size_t used = 0;
    for (const auto& alloc : pimpl->allocations) {
        if (alloc.second.device_id == device_id) {
            used += alloc.second.size;
        }
    }
    
    pimpl->device_memory_used[device_id] = used;
}

void* CUDAMemoryManager::allocate(size_t size, MemoryType type, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    void* ptr = nullptr;
    int device_id = getCurrentDevice();
    
    try {
        switch (type) {
            case MemoryType::HOST:
                ptr = allocateHostMemory(size);
                break;
            case MemoryType::DEVICE:
                checkMemoryLimit(size, device_id);
                ptr = allocateDeviceMemory(size, device_id);
                break;
            case MemoryType::MANAGED:
                checkMemoryLimit(size, device_id);
                ptr = allocateManagedMemory(size);
                break;
            case MemoryType::PINNED_HOST:
                ptr = allocatePinnedHostMemory(size);
                break;
            case MemoryType::SHARED:
                checkMemoryLimit(size, device_id);
                ptr = allocateManagedMemory(size);
                break;
            default:
                throw std::runtime_error("Invalid memory type");
        }
        
        // Record allocation
        MemoryAllocation alloc{
            ptr,
            size,
            type,
            device_id,
            false,
            tag
        };
        pimpl->allocations[ptr] = alloc;
        
        // Update memory stats
        if (type == MemoryType::DEVICE || type == MemoryType::MANAGED || type == MemoryType::SHARED) {
            pimpl->device_memory_used[device_id] += size;
        }
        
        spdlog::debug("Allocated {} bytes of {} memory on device {} with tag '{}'",
                     size, static_cast<int>(type), device_id, tag);
        
        return ptr;
    } catch (const std::exception& e) {
        spdlog::error("Failed to allocate memory: {}", e.what());
        throw;
    }
}

void CUDAMemoryManager::deallocate(void* ptr) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    auto it = pimpl->allocations.find(ptr);
    if (it == pimpl->allocations.end()) {
        throw std::runtime_error("Attempt to deallocate unknown pointer");
    }
    
    const auto& alloc = it->second;
    int device_id = alloc.device_id;
    
    try {
        switch (alloc.type) {
            case MemoryType::HOST:
                deallocateHostMemory(ptr);
                break;
            case MemoryType::DEVICE:
                deallocateDeviceMemory(ptr);
                break;
            case MemoryType::MANAGED:
                deallocateManagedMemory(ptr);
                break;
            case MemoryType::PINNED_HOST:
                deallocatePinnedHostMemory(ptr);
                break;
            case MemoryType::SHARED:
                deallocateManagedMemory(ptr);
                break;
        }
        
        // Update memory stats
        if (alloc.type == MemoryType::DEVICE || alloc.type == MemoryType::MANAGED || alloc.type == MemoryType::SHARED) {
            pimpl->device_memory_used[device_id] -= alloc.size;
        }
        
        // Remove allocation record
        pimpl->allocations.erase(it);
        
        spdlog::debug("Deallocated {} bytes of {} memory on device {} with tag '{}'",
                     alloc.size, static_cast<int>(alloc.type), device_id, alloc.tag);
    } catch (const std::exception& e) {
        spdlog::error("Failed to deallocate memory: {}", e.what());
        throw;
    }
}

void* CUDAMemoryManager::reallocate(void* ptr, size_t new_size) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    auto it = pimpl->allocations.find(ptr);
    if (it == pimpl->allocations.end()) {
        throw std::runtime_error("Attempt to reallocate unknown pointer");
    }
    
    const auto& old_alloc = it->second;
    int device_id = old_alloc.device_id;
    
    try {
        // Allocate new memory
        void* new_ptr = allocate(new_size, old_alloc.type, old_alloc.tag);
        
        // Copy data
        size_t copy_size = std::min(old_alloc.size, new_size);
        copy(new_ptr, ptr, copy_size, old_alloc.type, old_alloc.type);
        
        // Deallocate old memory
        deallocate(ptr);
        
        return new_ptr;
    } catch (const std::exception& e) {
        spdlog::error("Failed to reallocate memory: {}", e.what());
        throw;
    }
}

void CUDAMemoryManager::copy(void* dst, const void* src, size_t size, MemoryType dst_type, MemoryType src_type) {
    try {
        if (dst_type == MemoryType::DEVICE && src_type == MemoryType::HOST) {
            copyHostToDevice(dst, src, size);
        } else if (dst_type == MemoryType::HOST && src_type == MemoryType::DEVICE) {
            copyDeviceToHost(dst, src, size);
        } else if (dst_type == MemoryType::DEVICE && src_type == MemoryType::DEVICE) {
            copyDeviceToDevice(dst, src, size);
        } else if (dst_type == MemoryType::HOST && src_type == MemoryType::HOST) {
            copyHostToHost(dst, src, size);
        } else {
            throw std::runtime_error("Unsupported memory copy operation");
        }
    } catch (const std::exception& e) {
        spdlog::error("Failed to copy memory: {}", e.what());
        throw;
    }
}

void CUDAMemoryManager::memset(void* ptr, int value, size_t size, MemoryType type) {
    try {
        switch (type) {
            case MemoryType::HOST:
                memsetHost(ptr, value, size);
                break;
            case MemoryType::DEVICE:
                memsetDevice(ptr, value, size);
                break;
            case MemoryType::MANAGED:
                memsetManaged(ptr, value, size);
                break;
            default:
                throw std::runtime_error("Unsupported memory type for memset");
        }
    } catch (const std::exception& e) {
        spdlog::error("Failed to memset memory: {}", e.what());
        throw;
    }
}

void CUDAMemoryManager::prefetch(void* ptr, size_t size, int device_id) {
    try {
        prefetchToDevice(ptr, size, device_id);
    } catch (const std::exception& e) {
        spdlog::error("Failed to prefetch memory: {}", e.what());
        throw;
    }
}

size_t CUDAMemoryManager::getTotalMemory(int device_id) const {
    size_t free, total;
    getDeviceMemoryInfo(free, total, device_id);
    return total;
}

size_t CUDAMemoryManager::getFreeMemory(int device_id) const {
    size_t free, total;
    getDeviceMemoryInfo(free, total, device_id);
    return free;
}

size_t CUDAMemoryManager::getUsedMemory(int device_id) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    return pimpl->device_memory_used.at(device_id);
}

std::vector<MemoryAllocation> CUDAMemoryManager::getAllocations() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    std::vector<MemoryAllocation> result;
    result.reserve(pimpl->allocations.size());
    for (const auto& pair : pimpl->allocations) {
        result.push_back(pair.second);
    }
    return result;
}

MemoryAllocation CUDAMemoryManager::getAllocationInfo(void* ptr) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    auto it = pimpl->allocations.find(ptr);
    if (it == pimpl->allocations.end()) {
        throw std::runtime_error("Unknown pointer");
    }
    return it->second;
}

void CUDAMemoryManager::setMaxMemory(size_t max_memory, int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    pimpl->device_memory_limits[device_id] = max_memory;
}

void CUDAMemoryManager::clear() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Free all allocations
    for (const auto& pair : pimpl->allocations) {
        try {
            deallocate(pair.first);
        } catch (const std::exception& e) {
            spdlog::error("Failed to deallocate memory during clear: {}", e.what());
        }
    }
    
    pimpl->allocations.clear();
    pimpl->device_memory_used.clear();
}

void CUDAMemoryManager::reset() {
    clear();
    
    // Reinitialize all devices
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

// Helper function implementations
void* allocateHostMemory(size_t size) {
    void* ptr = nullptr;
    CUDA_CHECK(hipHostMalloc(&ptr, size));
    return ptr;
}

void* allocateDeviceMemory(size_t size, int device_id) {
    void* ptr = nullptr;
    CUDA_CHECK(hipSetDevice(device_id));
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

void* allocateManagedMemory(size_t size) {
    void* ptr = nullptr;
    CUDA_CHECK(hipMallocManaged(&ptr, size));
    return ptr;
}

void* allocatePinnedHostMemory(size_t size) {
    void* ptr = nullptr;
    CUDA_CHECK(hipHostMalloc(&ptr, size));
    return ptr;
}

void deallocateHostMemory(void* ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

void deallocateDeviceMemory(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
}

void deallocateManagedMemory(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
}

void deallocatePinnedHostMemory(void* ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

void copyHostToDevice(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void copyDeviceToHost(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void copyDeviceToDevice(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}

void copyHostToHost(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToHost));
}

void memsetHost(void* ptr, int value, size_t size) {
    CUDA_CHECK(cudaMemsetHost(ptr, value, size));
}

void memsetDevice(void* ptr, int value, size_t size) {
    CUDA_CHECK(hipMemset(ptr, value, size));
}

void memsetManaged(void* ptr, int value, size_t size) {
    CUDA_CHECK(hipMemset(ptr, value, size));
}

void prefetchToDevice(void* ptr, size_t size, int device_id) {
    CUDA_CHECK(hipMemPrefetchAsync(ptr, size, device_id));
}

void prefetchToHost(void* ptr, size_t size) {
    CUDA_CHECK(hipMemPrefetchAsync(ptr, size, hipCpuDeviceId));
}

} // namespace llm_inference
} // namespace msmartcompute 