#include "cuda_stream_manager.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>

namespace msmartcompute {
namespace llm_inference {

struct CUDAStreamManager::Impl {
    std::unordered_map<hipStream_t, StreamInfo> streams;
    std::mutex mutex;
    int num_devices;
};

CUDAStreamManager& CUDAStreamManager::getInstance() {
    static CUDAStreamManager instance;
    return instance;
}

CUDAStreamManager::CUDAStreamManager() : pimpl(std::make_unique<Impl>()) {
    CUDA_CHECK(hipGetDeviceCount(&pimpl->num_devices));
    for (int i = 0; i < pimpl->num_devices; ++i) {
        initializeDevice(i);
    }
}

CUDAStreamManager::~CUDAStreamManager() {
    clear();
}

void CUDAStreamManager::initializeDevice(int device_id) {
    CUDA_CHECK(hipSetDevice(device_id));
    spdlog::info("Initialized CUDA device {}", device_id);
}

void CUDAStreamManager::cleanupDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Destroy all streams associated with this device
    for (auto it = pimpl->streams.begin(); it != pimpl->streams.end();) {
        if (it->second.device_id == device_id) {
            CUDA_CHECK(hipStreamDestroy(it->first));
            it = pimpl->streams.erase(it);
        } else {
            ++it;
        }
    }
}

bool CUDAStreamManager::checkStream(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    return pimpl->streams.find(stream) != pimpl->streams.end();
}

hipStream_t CUDAStreamManager::createStream(StreamPriority priority, StreamFlags flags, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    int device_id;
    CUDA_CHECK(hipGetDevice(&device_id));
    
    hipStream_t stream;
    cudaStreamCreateFlags stream_flags = 0;
    if (flags == StreamFlags::NON_BLOCKING) {
        stream_flags = hipStreamNonBlocking;
    }
    
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, stream_flags));
    
    StreamInfo info;
    info.stream = stream;
    info.device_id = device_id;
    info.priority = priority;
    info.flags = flags;
    info.tag = tag;
    info.is_active = true;
    
    pimpl->streams[stream] = info;
    
    spdlog::debug("Created CUDA stream {} with tag '{}' on device {}", 
                 static_cast<void*>(stream), tag, device_id);
    
    return stream;
}

void CUDAStreamManager::destroyStream(hipStream_t stream) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to destroy unknown CUDA stream {}", static_cast<void*>(stream));
        return;
    }
    
    synchronize(stream);
    CUDA_CHECK(hipStreamDestroy(stream));
    pimpl->streams.erase(stream);
    
    spdlog::debug("Destroyed CUDA stream {}", static_cast<void*>(stream));
}

void CUDAStreamManager::synchronize(hipStream_t stream) {
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to synchronize unknown CUDA stream {}", static_cast<void*>(stream));
        return;
    }
    
    CUDA_CHECK(hipStreamSynchronize(stream));
}

void CUDAStreamManager::synchronizeAll() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    for (const auto& [stream, info] : pimpl->streams) {
        if (info.is_active) {
            CUDA_CHECK(hipStreamSynchronize(stream));
        }
    }
}

bool CUDAStreamManager::isStreamActive(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        return false;
    }
    
    return pimpl->streams.at(stream).is_active;
}

void CUDAStreamManager::setStreamActive(hipStream_t stream, bool active) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to set active state for unknown CUDA stream {}", 
                    static_cast<void*>(stream));
        return;
    }
    
    pimpl->streams[stream].is_active = active;
}

StreamInfo CUDAStreamManager::getStreamInfo(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        throw std::runtime_error("Unknown CUDA stream");
    }
    
    return pimpl->streams.at(stream);
}

std::vector<StreamInfo> CUDAStreamManager::getAllStreams() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    std::vector<StreamInfo> result;
    result.reserve(pimpl->streams.size());
    
    for (const auto& [stream, info] : pimpl->streams) {
        result.push_back(info);
    }
    
    return result;
}

size_t CUDAStreamManager::getStreamCount() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    return pimpl->streams.size();
}

void CUDAStreamManager::setStreamPriority(hipStream_t stream, StreamPriority priority) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to set priority for unknown CUDA stream {}", 
                    static_cast<void*>(stream));
        return;
    }
    
    pimpl->streams[stream].priority = priority;
}

void CUDAStreamManager::setStreamFlags(hipStream_t stream, StreamFlags flags) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to set flags for unknown CUDA stream {}", 
                    static_cast<void*>(stream));
        return;
    }
    
    pimpl->streams[stream].flags = flags;
}

void CUDAStreamManager::setStreamTag(hipStream_t stream, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (!checkStream(stream)) {
        spdlog::warn("Attempted to set tag for unknown CUDA stream {}", 
                    static_cast<void*>(stream));
        return;
    }
    
    pimpl->streams[stream].tag = tag;
}

void CUDAStreamManager::clear() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    for (const auto& [stream, info] : pimpl->streams) {
        CUDA_CHECK(hipStreamDestroy(stream));
    }
    
    pimpl->streams.clear();
}

void CUDAStreamManager::reset() {
    clear();
    
    for (int i = 0; i < pimpl->num_devices; ++i) {
        initializeDevice(i);
    }
}

} // namespace llm_inference
} // namespace msmartcompute 