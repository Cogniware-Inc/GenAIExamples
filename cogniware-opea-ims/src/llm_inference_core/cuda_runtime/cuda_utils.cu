#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>

namespace msmartcompute {
namespace llm_inference {

CUDADeviceProperties getDeviceProperties(int device_id) {
    CUDADeviceProperties props;
    props.device_id = device_id;

    // Get device properties
    CUDA_CHECK(hipGetDeviceProperties(&props.properties, device_id));

    // Get memory info
    size_t free, total;
    getDeviceMemoryInfo(free, total, device_id);
    props.total_memory = total;
    props.free_memory = free;

    // Get compute capability
    props.compute_capability_major = props.properties.major;
    props.compute_capability_minor = props.properties.minor;

    // Get multi processor count
    props.multi_processor_count = props.properties.multiProcessorCount;

    // Get max threads per block
    props.max_threads_per_block = props.properties.maxThreadsPerBlock;

    // Get warp size
    props.warp_size = props.properties.warpSize;

    // Get max shared memory per block
    props.max_shared_memory_per_block = props.properties.sharedMemPerBlock;

    // Get max registers per block
    props.max_registers_per_block = props.properties.regsPerBlock;

    // Get max threads per multi processor
    props.max_threads_per_multi_processor = props.properties.maxThreadsPerMultiProcessor;

    // Get max blocks per multi processor
    props.max_blocks_per_multi_processor = props.properties.maxBlocksPerMultiProcessor;

    // Get max grid dimensions
    props.max_grid_dim_x = props.properties.maxGridSize[0];
    props.max_grid_dim_y = props.properties.maxGridSize[1];
    props.max_grid_dim_z = props.properties.maxGridSize[2];

    // Get max block dimensions
    props.max_block_dim_x = props.properties.maxThreadsDim[0];
    props.max_block_dim_y = props.properties.maxThreadsDim[1];
    props.max_block_dim_z = props.properties.maxThreadsDim[2];

    // Get clock rate
    props.clock_rate = props.properties.clockRate;

    // Get memory clock rate
    props.memory_clock_rate = props.properties.memoryClockRate;

    // Get memory bus width
    props.memory_bus_width = props.properties.memoryBusWidth;

    // Get L2 cache size
    props.l2_cache_size = props.properties.l2CacheSize;

    // Calculate derived properties
    props.max_threads_per_sm = props.properties.maxThreadsPerMultiProcessor;
    props.max_blocks_per_sm = props.properties.maxBlocksPerMultiProcessor;
    props.max_shared_memory_per_sm = props.properties.sharedMemPerMultiprocessor;
    props.max_registers_per_sm = props.properties.regsPerMultiprocessor;
    props.max_warps_per_sm = props.max_threads_per_sm / props.warp_size;
    props.max_threads_per_warp = props.warp_size;
    props.max_blocks_per_grid = props.max_grid_dim_x * props.max_grid_dim_y * props.max_grid_dim_z;
    props.max_shared_memory_per_grid = props.max_shared_memory_per_block * props.max_blocks_per_grid;
    props.max_registers_per_grid = props.max_registers_per_block * props.max_blocks_per_grid;
    props.max_warps_per_grid = props.max_threads_per_grid / props.warp_size;
    props.max_threads_per_grid = props.max_threads_per_block * props.max_blocks_per_grid;
    props.max_blocks_per_device = props.max_blocks_per_sm * props.multi_processor_count;
    props.max_shared_memory_per_device = props.max_shared_memory_per_sm * props.multi_processor_count;
    props.max_registers_per_device = props.max_registers_per_sm * props.multi_processor_count;
    props.max_warps_per_device = props.max_warps_per_sm * props.multi_processor_count;
    props.max_threads_per_device = props.max_threads_per_sm * props.multi_processor_count;

    return props;
}

void initializeCUDA(int device_id) {
    try {
        // Set device
        CUDA_CHECK(hipSetDevice(device_id));

        // Get device properties
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));

        spdlog::info("Initialized CUDA device {}: {}", device_id, prop.name);
        spdlog::info("Compute capability: {}.{}", prop.major, prop.minor);
        spdlog::info("Total memory: {} MB", prop.totalGlobalMem / (1024 * 1024));
        spdlog::info("Multi-processors: {}", prop.multiProcessorCount);
        spdlog::info("Max threads per block: {}", prop.maxThreadsPerBlock);
        spdlog::info("Warp size: {}", prop.warpSize);
    } catch (const std::exception& e) {
        spdlog::error("Failed to initialize CUDA: {}", e.what());
        throw;
    }
}

int getCurrentDevice() {
    int device;
    CUDA_CHECK(hipGetDevice(&device));
    return device;
}

void setDevice(int device_id) {
    CUDA_CHECK(hipSetDevice(device_id));
}

int getDeviceCount() {
    int count;
    CUDA_CHECK(hipGetDeviceCount(&count));
    return count;
}

std::string getDeviceName(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.name;
}

void getDeviceMemoryInfo(size_t& free, size_t& total, int device_id) {
    CUDA_CHECK(hipSetDevice(device_id));
    CUDA_CHECK(hipMemGetInfo(&free, &total));
}

void getDeviceComputeCapability(int& major, int& minor, int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    major = prop.major;
    minor = prop.minor;
}

int getDeviceMultiProcessorCount(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.multiProcessorCount;
}

int getDeviceMaxThreadsPerBlock(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerBlock;
}

int getDeviceWarpSize(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.warpSize;
}

int getDeviceMaxSharedMemoryPerBlock(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.sharedMemPerBlock;
}

int getDeviceMaxRegistersPerBlock(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.regsPerBlock;
}

int getDeviceMaxThreadsPerMultiProcessor(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerMultiProcessor;
}

int getDeviceMaxBlocksPerMultiProcessor(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxBlocksPerMultiProcessor;
}

void getDeviceMaxGridDimensions(int& x, int& y, int& z, int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    x = prop.maxGridSize[0];
    y = prop.maxGridSize[1];
    z = prop.maxGridSize[2];
}

void getDeviceMaxBlockDimensions(int& x, int& y, int& z, int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    x = prop.maxThreadsDim[0];
    y = prop.maxThreadsDim[1];
    z = prop.maxThreadsDim[2];
}

int getDeviceClockRate(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.clockRate;
}

int getDeviceMemoryClockRate(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.memoryClockRate;
}

int getDeviceMemoryBusWidth(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.memoryBusWidth;
}

int getDeviceL2CacheSize(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.l2CacheSize;
}

int getDeviceMaxThreadsPerSM(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerMultiProcessor;
}

int getDeviceMaxBlocksPerSM(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxBlocksPerMultiProcessor;
}

int getDeviceMaxSharedMemoryPerSM(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.sharedMemPerMultiprocessor;
}

int getDeviceMaxRegistersPerSM(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.regsPerMultiprocessor;
}

int getDeviceMaxWarpsPerSM(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerMultiProcessor / prop.warpSize;
}

int getDeviceMaxThreadsPerWarp(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.warpSize;
}

int getDeviceMaxBlocksPerGrid(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxGridSize[0] * prop.maxGridSize[1] * prop.maxGridSize[2];
}

int getDeviceMaxSharedMemoryPerGrid(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.sharedMemPerBlock * getDeviceMaxBlocksPerGrid(device_id);
}

int getDeviceMaxRegistersPerGrid(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.regsPerBlock * getDeviceMaxBlocksPerGrid(device_id);
}

int getDeviceMaxWarpsPerGrid(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return getDeviceMaxThreadsPerGrid(device_id) / prop.warpSize;
}

int getDeviceMaxThreadsPerGrid(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerBlock * getDeviceMaxBlocksPerGrid(device_id);
}

int getDeviceMaxBlocksPerDevice(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxBlocksPerMultiProcessor * prop.multiProcessorCount;
}

int getDeviceMaxSharedMemoryPerDevice(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.sharedMemPerMultiprocessor * prop.multiProcessorCount;
}

int getDeviceMaxRegistersPerDevice(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.regsPerMultiprocessor * prop.multiProcessorCount;
}

int getDeviceMaxWarpsPerDevice(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return getDeviceMaxThreadsPerDevice(device_id) / prop.warpSize;
}

int getDeviceMaxThreadsPerDevice(int device_id) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
    return prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
}

} // namespace llm_inference
} // namespace msmartcompute
