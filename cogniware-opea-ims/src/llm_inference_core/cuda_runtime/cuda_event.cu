#include "cuda_event.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>
#include <unordered_map>
#include <mutex>
#include <stdexcept>

namespace msmartcompute {
namespace llm_inference {

// Implementation of CUDAEventManager
struct CUDAEventManager::Impl {
    std::unordered_map<hipEvent_t, EventInfo> events;
    std::mutex mutex;
};

CUDAEventManager& CUDAEventManager::getInstance() {
    static CUDAEventManager instance;
    return instance;
}

CUDAEventManager::CUDAEventManager() : pimpl(std::make_unique<Impl>()) {
    // Initialize event manager
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

CUDAEventManager::~CUDAEventManager() {
    clear();
}

void CUDAEventManager::initializeDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Set device
    setDevice(device_id);
    
    spdlog::info("Initialized CUDA event manager for device {}", device_id);
}

void CUDAEventManager::cleanupDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Destroy all events for this device
    for (auto it = pimpl->events.begin(); it != pimpl->events.end();) {
        if (it->second.device_id == device_id) {
            destroyEvent(it->first);
            it = pimpl->events.erase(it);
        } else {
            ++it;
        }
    }
}

void CUDAEventManager::checkEvent(hipEvent_t event) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (pimpl->events.find(event) == pimpl->events.end()) {
        throw std::runtime_error("Unknown event");
    }
}

hipEvent_t CUDAEventManager::createEvent(EventFlags flags, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    int device_id = getCurrentDevice();
    
    try {
        // Create event
        hipEvent_t event;
        CUDA_CHECK(hipEventCreateWithFlags(&event, static_cast<cudaEventFlags>(flags)));
        
        // Record event info
        EventInfo info{
            event,
            device_id,
            flags,
            tag,
            true,
            0.0f
        };
        pimpl->events[event] = info;
        
        spdlog::debug("Created CUDA event on device {} with flags {} and tag '{}'",
                     device_id, static_cast<int>(flags), tag);
        
        return event;
    } catch (const std::exception& e) {
        spdlog::error("Failed to create CUDA event: {}", e.what());
        throw;
    }
}

void CUDAEventManager::destroyEvent(hipEvent_t event) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    auto it = pimpl->events.find(event);
    if (it == pimpl->events.end()) {
        throw std::runtime_error("Attempt to destroy unknown event");
    }
    
    try {
        // Synchronize event before destroying
        synchronize(event);
        
        // Destroy event
        CUDA_CHECK(hipEventDestroy(event));
        
        // Remove event info
        pimpl->events.erase(it);
        
        spdlog::debug("Destroyed CUDA event with tag '{}'", it->second.tag);
    } catch (const std::exception& e) {
        spdlog::error("Failed to destroy CUDA event: {}", e.what());
        throw;
    }
}

void CUDAEventManager::record(hipEvent_t event, hipStream_t stream) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    
    try {
        CUDA_CHECK(hipEventRecord(event, stream));
    } catch (const std::exception& e) {
        spdlog::error("Failed to record CUDA event: {}", e.what());
        throw;
    }
}

void CUDAEventManager::synchronize(hipEvent_t event) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    
    try {
        CUDA_CHECK(hipEventSynchronize(event));
    } catch (const std::exception& e) {
        spdlog::error("Failed to synchronize CUDA event: {}", e.what());
        throw;
    }
}

float CUDAEventManager::getElapsedTime(hipEvent_t start, hipEvent_t end) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(start);
    checkEvent(end);
    
    try {
        float elapsed_time;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, end));
        return elapsed_time;
    } catch (const std::exception& e) {
        spdlog::error("Failed to get elapsed time between events: {}", e.what());
        throw;
    }
}

bool CUDAEventManager::isEventActive(hipEvent_t event) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    return pimpl->events.at(event).is_active;
}

void CUDAEventManager::setEventActive(hipEvent_t event, bool active) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    pimpl->events[event].is_active = active;
}

EventInfo CUDAEventManager::getEventInfo(hipEvent_t event) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    return pimpl->events.at(event);
}

std::vector<EventInfo> CUDAEventManager::getAllEvents() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    std::vector<EventInfo> result;
    result.reserve(pimpl->events.size());
    for (const auto& pair : pimpl->events) {
        result.push_back(pair.second);
    }
    return result;
}

int CUDAEventManager::getEventCount() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    return static_cast<int>(pimpl->events.size());
}

void CUDAEventManager::setEventFlags(hipEvent_t event, EventFlags flags) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    pimpl->events[event].flags = flags;
}

void CUDAEventManager::setEventTag(hipEvent_t event, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkEvent(event);
    pimpl->events[event].tag = tag;
}

void CUDAEventManager::clear() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Destroy all events
    for (const auto& pair : pimpl->events) {
        try {
            destroyEvent(pair.first);
        } catch (const std::exception& e) {
            spdlog::error("Failed to destroy event during clear: {}", e.what());
        }
    }
    
    pimpl->events.clear();
}

void CUDAEventManager::reset() {
    clear();
    
    // Reinitialize all devices
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

// Helper function implementations
hipEvent_t createEvent(EventFlags flags) {
    return CUDAEventManager::getInstance().createEvent(flags);
}

void destroyEvent(hipEvent_t event) {
    CUDAEventManager::getInstance().destroyEvent(event);
}

void recordEvent(hipEvent_t event, hipStream_t stream) {
    CUDAEventManager::getInstance().record(event, stream);
}

void synchronizeEvent(hipEvent_t event) {
    CUDAEventManager::getInstance().synchronize(event);
}

float getElapsedTime(hipEvent_t start, hipEvent_t end) {
    return CUDAEventManager::getInstance().getElapsedTime(start, end);
}

bool isEventActive(hipEvent_t event) {
    return CUDAEventManager::getInstance().isEventActive(event);
}

void setEventActive(hipEvent_t event, bool active) {
    CUDAEventManager::getInstance().setEventActive(event, active);
}

void setEventFlags(hipEvent_t event, EventFlags flags) {
    CUDAEventManager::getInstance().setEventFlags(event, flags);
}

} // namespace llm_inference
} // namespace msmartcompute 