#include "hip_stream.h"
#include "cuda_utils.h"
#include <spdlog/spdlog.h>
#include <unordered_map>
#include <mutex>
#include <stdexcept>

namespace msmartcompute {
namespace llm_inference {

// Implementation of CUDAStreamManager
struct CUDAStreamManager::Impl {
    std::unordered_map<hipStream_t, StreamInfo> streams;
    std::mutex mutex;
};

CUDAStreamManager& CUDAStreamManager::getInstance() {
    static CUDAStreamManager instance;
    return instance;
}

CUDAStreamManager::CUDAStreamManager() : pimpl(std::make_unique<Impl>()) {
    // Initialize stream manager
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

CUDAStreamManager::~CUDAStreamManager() {
    clear();
}

void CUDAStreamManager::initializeDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Set device
    setDevice(device_id);
    
    spdlog::info("Initialized CUDA stream manager for device {}", device_id);
}

void CUDAStreamManager::cleanupDevice(int device_id) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Destroy all streams for this device
    for (auto it = pimpl->streams.begin(); it != pimpl->streams.end();) {
        if (it->second.device_id == device_id) {
            destroyStream(it->first);
            it = pimpl->streams.erase(it);
        } else {
            ++it;
        }
    }
}

void CUDAStreamManager::checkStream(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    if (pimpl->streams.find(stream) == pimpl->streams.end()) {
        throw std::runtime_error("Unknown stream");
    }
}

hipStream_t CUDAStreamManager::createStream(StreamPriority priority, StreamFlags flags, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    int device_id = getCurrentDevice();
    
    try {
        // Create stream
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreateWithPriority(&stream, static_cast<cudaStreamFlags>(flags), static_cast<int>(priority)));
        
        // Record stream info
        StreamInfo info{
            stream,
            device_id,
            priority,
            flags,
            tag,
            true
        };
        pimpl->streams[stream] = info;
        
        spdlog::debug("Created CUDA stream on device {} with priority {} and tag '{}'",
                     device_id, static_cast<int>(priority), tag);
        
        return stream;
    } catch (const std::exception& e) {
        spdlog::error("Failed to create CUDA stream: {}", e.what());
        throw;
    }
}

void CUDAStreamManager::destroyStream(hipStream_t stream) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    auto it = pimpl->streams.find(stream);
    if (it == pimpl->streams.end()) {
        throw std::runtime_error("Attempt to destroy unknown stream");
    }
    
    try {
        // Synchronize stream before destroying
        synchronize(stream);
        
        // Destroy stream
        CUDA_CHECK(hipStreamDestroy(stream));
        
        // Remove stream info
        pimpl->streams.erase(it);
        
        spdlog::debug("Destroyed CUDA stream with tag '{}'", it->second.tag);
    } catch (const std::exception& e) {
        spdlog::error("Failed to destroy CUDA stream: {}", e.what());
        throw;
    }
}

void CUDAStreamManager::synchronize(hipStream_t stream) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    
    try {
        CUDA_CHECK(hipStreamSynchronize(stream));
    } catch (const std::exception& e) {
        spdlog::error("Failed to synchronize CUDA stream: {}", e.what());
        throw;
    }
}

void CUDAStreamManager::synchronizeAll() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    try {
        for (const auto& pair : pimpl->streams) {
            if (pair.second.is_active) {
                CUDA_CHECK(hipStreamSynchronize(pair.first));
            }
        }
    } catch (const std::exception& e) {
        spdlog::error("Failed to synchronize all CUDA streams: {}", e.what());
        throw;
    }
}

bool CUDAStreamManager::isStreamActive(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    return pimpl->streams.at(stream).is_active;
}

void CUDAStreamManager::setStreamActive(hipStream_t stream, bool active) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    pimpl->streams[stream].is_active = active;
}

StreamInfo CUDAStreamManager::getStreamInfo(hipStream_t stream) const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    return pimpl->streams.at(stream);
}

std::vector<StreamInfo> CUDAStreamManager::getAllStreams() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    std::vector<StreamInfo> result;
    result.reserve(pimpl->streams.size());
    for (const auto& pair : pimpl->streams) {
        result.push_back(pair.second);
    }
    return result;
}

int CUDAStreamManager::getStreamCount() const {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    return static_cast<int>(pimpl->streams.size());
}

void CUDAStreamManager::setStreamPriority(hipStream_t stream, StreamPriority priority) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    
    try {
        CUDA_CHECK(cudaStreamSetPriority(stream, static_cast<int>(priority)));
        pimpl->streams[stream].priority = priority;
    } catch (const std::exception& e) {
        spdlog::error("Failed to set stream priority: {}", e.what());
        throw;
    }
}

void CUDAStreamManager::setStreamFlags(hipStream_t stream, StreamFlags flags) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    pimpl->streams[stream].flags = flags;
}

void CUDAStreamManager::setStreamTag(hipStream_t stream, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    checkStream(stream);
    pimpl->streams[stream].tag = tag;
}

void CUDAStreamManager::clear() {
    std::lock_guard<std::mutex> lock(pimpl->mutex);
    
    // Destroy all streams
    for (const auto& pair : pimpl->streams) {
        try {
            destroyStream(pair.first);
        } catch (const std::exception& e) {
            spdlog::error("Failed to destroy stream during clear: {}", e.what());
        }
    }
    
    pimpl->streams.clear();
}

void CUDAStreamManager::reset() {
    clear();
    
    // Reinitialize all devices
    int device_count = getDeviceCount();
    for (int i = 0; i < device_count; ++i) {
        initializeDevice(i);
    }
}

// Helper function implementations
hipStream_t createStream(StreamPriority priority, StreamFlags flags) {
    return CUDAStreamManager::getInstance().createStream(priority, flags);
}

void destroyStream(hipStream_t stream) {
    CUDAStreamManager::getInstance().destroyStream(stream);
}

void synchronizeStream(hipStream_t stream) {
    CUDAStreamManager::getInstance().synchronize(stream);
}

void synchronizeAllStreams() {
    CUDAStreamManager::getInstance().synchronizeAll();
}

bool isStreamActive(hipStream_t stream) {
    return CUDAStreamManager::getInstance().isStreamActive(stream);
}

void setStreamActive(hipStream_t stream, bool active) {
    CUDAStreamManager::getInstance().setStreamActive(stream, active);
}

void setStreamPriority(hipStream_t stream, StreamPriority priority) {
    CUDAStreamManager::getInstance().setStreamPriority(stream, priority);
}

void setStreamFlags(hipStream_t stream, StreamFlags flags) {
    CUDAStreamManager::getInstance().setStreamFlags(stream, flags);
}

// Stream callback implementation
void CUDART_CB streamCallback(hipStream_t stream, hipError_t status, void* userData) {
    auto* callback = static_cast<StreamCallback*>(userData);
    (*callback)(stream, status, nullptr);
    delete callback;
}

void addStreamCallback(hipStream_t stream, StreamCallback callback, void* userData) {
    auto* callback_ptr = new StreamCallback(std::move(callback));
    CUDA_CHECK(hipStreamAddCallback(stream, streamCallback, callback_ptr, 0));
}

} // namespace llm_inference
} // namespace msmartcompute 