#include "hip/hip_runtime.h"
#include "matrix_vector_ops.h"
#include "cuda_utils.h"
#include <hipblas.h>
#include <spdlog/spdlog.h>

namespace msmartcompute {
namespace llm_inference {

// Helper function to get cuBLAS handle
static hipblasHandle_t getCublasHandle() {
    static hipblasHandle_t handle = nullptr;
    if (handle == nullptr) {
        CUDA_CHECK(hipblasCreate(&handle));
    }
    return handle;
}

// Matrix-vector multiplication kernels
__global__ void matrixVectorMultiplyKernel(
    float* output,
    const float* matrix,
    const float* vector,
    int rows,
    int cols,
    float alpha,
    float beta
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0.0f;
        for (int col = 0; col < cols; ++col) {
            sum += matrix[row * cols + col] * vector[col];
        }
        output[row] = alpha * sum + beta * output[row];
    }
}

__global__ void matrixVectorMultiplyKernel(
    half* output,
    const half* matrix,
    const half* vector,
    int rows,
    int cols,
    float alpha,
    float beta
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0.0f;
        for (int col = 0; col < cols; ++col) {
            sum += __half2float(matrix[row * cols + col]) * __half2float(vector[col]);
        }
        output[row] = __float2half(alpha * sum + beta * __half2float(output[row]));
    }
}

// Matrix-vector multiplication implementation
void matrixVectorMultiply(
    float* output,
    const float* matrix,
    const float* vector,
    int rows,
    int cols,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(hipblasSgemv(
        handle,
        HIPBLAS_OP_N,
        rows,
        cols,
        alpha_ptr,
        matrix,
        rows,
        vector,
        1,
        beta_ptr,
        output,
        1
    ));
}

void matrixVectorMultiply(
    half* output,
    const half* matrix,
    const half* vector,
    int rows,
    int cols,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(cublasHgemv(
        handle,
        HIPBLAS_OP_N,
        rows,
        cols,
        alpha_ptr,
        matrix,
        rows,
        vector,
        1,
        beta_ptr,
        output,
        1
    ));
}

// Matrix-matrix multiplication implementation
void matrixMultiply(
    float* output,
    const float* matrix_a,
    const float* matrix_b,
    int m,
    int k,
    int n,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(hipblasSgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        alpha_ptr,
        matrix_a,
        m,
        matrix_b,
        k,
        beta_ptr,
        output,
        m
    ));
}

void matrixMultiply(
    half* output,
    const half* matrix_a,
    const half* matrix_b,
    int m,
    int k,
    int n,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(hipblasHgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        alpha_ptr,
        matrix_a,
        m,
        matrix_b,
        k,
        beta_ptr,
        output,
        m
    ));
}

// Vector operations kernels
__global__ void vectorAddKernel(
    float* output,
    const float* vector_a,
    const float* vector_b,
    int size,
    float alpha,
    float beta
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = alpha * vector_a[idx] + beta * vector_b[idx];
    }
}

__global__ void vectorAddKernel(
    half* output,
    const half* vector_a,
    const half* vector_b,
    int size,
    float alpha,
    float beta
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(
            alpha * __half2float(vector_a[idx]) +
            beta * __half2float(vector_b[idx])
        );
    }
}

__global__ void vectorScaleKernel(
    float* output,
    const float* vector,
    int size,
    float scale
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = scale * vector[idx];
    }
}

__global__ void vectorScaleKernel(
    half* output,
    const half* vector,
    int size,
    float scale
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(scale * __half2float(vector[idx]));
    }
}

// Vector operations implementation
void vectorAdd(
    float* output,
    const float* vector_a,
    const float* vector_b,
    int size,
    float alpha,
    float beta,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    vectorAddKernel<<<num_blocks, block_size, 0, stream>>>(
        output, vector_a, vector_b, size, alpha, beta
    );
    CUDA_CHECK(hipGetLastError());
}

void vectorAdd(
    half* output,
    const half* vector_a,
    const half* vector_b,
    int size,
    float alpha,
    float beta,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    vectorAddKernel<<<num_blocks, block_size, 0, stream>>>(
        output, vector_a, vector_b, size, alpha, beta
    );
    CUDA_CHECK(hipGetLastError());
}

void vectorScale(
    float* output,
    const float* vector,
    int size,
    float scale,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    vectorScaleKernel<<<num_blocks, block_size, 0, stream>>>(
        output, vector, size, scale
    );
    CUDA_CHECK(hipGetLastError());
}

void vectorScale(
    half* output,
    const half* vector,
    int size,
    float scale,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    
    vectorScaleKernel<<<num_blocks, block_size, 0, stream>>>(
        output, vector, size, scale
    );
    CUDA_CHECK(hipGetLastError());
}

// Matrix operations kernels
__global__ void matrixAddKernel(
    float* output,
    const float* matrix_a,
    const float* matrix_b,
    int rows,
    int cols,
    float alpha,
    float beta
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        output[idx] = alpha * matrix_a[idx] + beta * matrix_b[idx];
    }
}

__global__ void matrixAddKernel(
    half* output,
    const half* matrix_a,
    const half* matrix_b,
    int rows,
    int cols,
    float alpha,
    float beta
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        output[idx] = __float2half(
            alpha * __half2float(matrix_a[idx]) +
            beta * __half2float(matrix_b[idx])
        );
    }
}

__global__ void matrixScaleKernel(
    float* output,
    const float* matrix,
    int rows,
    int cols,
    float scale
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        output[idx] = scale * matrix[idx];
    }
}

__global__ void matrixScaleKernel(
    half* output,
    const half* matrix,
    int rows,
    int cols,
    float scale
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        output[idx] = __float2half(scale * __half2float(matrix[idx]));
    }
}

// Matrix operations implementation
void matrixAdd(
    float* output,
    const float* matrix_a,
    const float* matrix_b,
    int rows,
    int cols,
    float alpha,
    float beta,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixAddKernel<<<grid, block, 0, stream>>>(
        output, matrix_a, matrix_b, rows, cols, alpha, beta
    );
    CUDA_CHECK(hipGetLastError());
}

void matrixAdd(
    half* output,
    const half* matrix_a,
    const half* matrix_b,
    int rows,
    int cols,
    float alpha,
    float beta,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixAddKernel<<<grid, block, 0, stream>>>(
        output, matrix_a, matrix_b, rows, cols, alpha, beta
    );
    CUDA_CHECK(hipGetLastError());
}

void matrixScale(
    float* output,
    const float* matrix,
    int rows,
    int cols,
    float scale,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixScaleKernel<<<grid, block, 0, stream>>>(
        output, matrix, rows, cols, scale
    );
    CUDA_CHECK(hipGetLastError());
}

void matrixScale(
    half* output,
    const half* matrix,
    int rows,
    int cols,
    float scale,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixScaleKernel<<<grid, block, 0, stream>>>(
        output, matrix, rows, cols, scale
    );
    CUDA_CHECK(hipGetLastError());
}

// Transpose operations kernels
__global__ void matrixTransposeKernel(
    float* output,
    const float* input,
    int rows,
    int cols
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        output[col * rows + row] = input[row * cols + col];
    }
}

__global__ void matrixTransposeKernel(
    half* output,
    const half* input,
    int rows,
    int cols
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        output[col * rows + row] = input[row * cols + col];
    }
}

// Transpose operations implementation
void matrixTranspose(
    float* output,
    const float* input,
    int rows,
    int cols,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixTransposeKernel<<<grid, block, 0, stream>>>(
        output, input, rows, cols
    );
    CUDA_CHECK(hipGetLastError());
}

void matrixTranspose(
    half* output,
    const half* input,
    int rows,
    int cols,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    matrixTransposeKernel<<<grid, block, 0, stream>>>(
        output, input, rows, cols
    );
    CUDA_CHECK(hipGetLastError());
}

// Batch matrix multiplication implementation
void batchMatrixMultiply(
    float* output,
    const float* matrix_a,
    const float* matrix_b,
    int batch_size,
    int m,
    int k,
    int n,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(hipblasSgemmStridedBatched(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        alpha_ptr,
        matrix_a,
        m,
        m * k,
        matrix_b,
        k,
        k * n,
        beta_ptr,
        output,
        m,
        m * n,
        batch_size
    ));
}

void batchMatrixMultiply(
    half* output,
    const half* matrix_a,
    const half* matrix_b,
    int batch_size,
    int m,
    int k,
    int n,
    float alpha,
    float beta,
    hipStream_t stream
) {
    hipblasHandle_t handle = getCublasHandle();
    if (stream) {
        CUDA_CHECK(hipblasSetStream(handle, stream));
    }
    
    const float* alpha_ptr = &alpha;
    const float* beta_ptr = &beta;
    
    CUDA_CHECK(hipblasHgemmStridedBatched(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        alpha_ptr,
        matrix_a,
        m,
        m * k,
        matrix_b,
        k,
        k * n,
        beta_ptr,
        output,
        m,
        m * n,
        batch_size
    ));
}

// Reduction operations kernels
__global__ void reduceSumKernel(
    float* output,
    const float* input,
    int size
) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (idx < size) ? input[idx] : 0.0f;
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

__global__ void reduceSumKernel(
    half* output,
    const half* input,
    int size
) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (idx < size) ? __half2float(input[idx]) : 0.0f;
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = __float2half(sdata[0]);
    }
}

__global__ void reduceMaxKernel(
    float* output,
    const float* input,
    int size
) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (idx < size) ? input[idx] : -INFINITY;
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] = max(sdata[tid], sdata[tid + offset]);
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

__global__ void reduceMaxKernel(
    half* output,
    const half* input,
    int size
) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (idx < size) ? __half2float(input[idx]) : -INFINITY;
    __syncthreads();
    
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] = max(sdata[tid], sdata[tid + offset]);
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = __float2half(sdata[0]);
    }
}

// Reduction operations implementation
void reduceSum(
    float* output,
    const float* input,
    int size,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    size_t shared_mem_size = block_size * sizeof(float);
    
    reduceSumKernel<<<num_blocks, block_size, shared_mem_size, stream>>>(
        output, input, size
    );
    CUDA_CHECK(hipGetLastError());
}

void reduceSum(
    half* output,
    const half* input,
    int size,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    size_t shared_mem_size = block_size * sizeof(float);
    
    reduceSumKernel<<<num_blocks, block_size, shared_mem_size, stream>>>(
        output, input, size
    );
    CUDA_CHECK(hipGetLastError());
}

void reduceMax(
    float* output,
    const float* input,
    int size,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    size_t shared_mem_size = block_size * sizeof(float);
    
    reduceMaxKernel<<<num_blocks, block_size, shared_mem_size, stream>>>(
        output, input, size
    );
    CUDA_CHECK(hipGetLastError());
}

void reduceMax(
    half* output,
    const half* input,
    int size,
    hipStream_t stream
) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    size_t shared_mem_size = block_size * sizeof(float);
    
    reduceMaxKernel<<<num_blocks, block_size, shared_mem_size, stream>>>(
        output, input, size
    );
    CUDA_CHECK(hipGetLastError());
}

// Utility functions kernels
__global__ void setMatrixToIdentityKernel(
    float* matrix,
    int size
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < size && col < size) {
        matrix[row * size + col] = (row == col) ? 1.0f : 0.0f;
    }
}

__global__ void setMatrixToIdentityKernel(
    half* matrix,
    int size
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < size && col < size) {
        matrix[row * size + col] = (row == col) ? __float2half(1.0f) : __float2half(0.0f);
    }
}

__global__ void setMatrixToZeroKernel(
    float* matrix,
    int rows,
    int cols
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        matrix[row * cols + col] = 0.0f;
    }
}

__global__ void setMatrixToZeroKernel(
    half* matrix,
    int rows,
    int cols
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        matrix[row * cols + col] = __float2half(0.0f);
    }
}

// Utility functions implementation
void setMatrixToIdentity(
    float* matrix,
    int size,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (size + block.x - 1) / block.x,
        (size + block.y - 1) / block.y
    );
    
    setMatrixToIdentityKernel<<<grid, block, 0, stream>>>(
        matrix, size
    );
    CUDA_CHECK(hipGetLastError());
}

void setMatrixToIdentity(
    half* matrix,
    int size,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (size + block.x - 1) / block.x,
        (size + block.y - 1) / block.y
    );
    
    setMatrixToIdentityKernel<<<grid, block, 0, stream>>>(
        matrix, size
    );
    CUDA_CHECK(hipGetLastError());
}

void setMatrixToZero(
    float* matrix,
    int rows,
    int cols,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    setMatrixToZeroKernel<<<grid, block, 0, stream>>>(
        matrix, rows, cols
    );
    CUDA_CHECK(hipGetLastError());
}

void setMatrixToZero(
    half* matrix,
    int rows,
    int cols,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid(
        (rows + block.x - 1) / block.x,
        (cols + block.y - 1) / block.y
    );
    
    setMatrixToZeroKernel<<<grid, block, 0, stream>>>(
        matrix, rows, cols
    );
    CUDA_CHECK(hipGetLastError());
}

} // namespace llm_inference
} // namespace msmartcompute
