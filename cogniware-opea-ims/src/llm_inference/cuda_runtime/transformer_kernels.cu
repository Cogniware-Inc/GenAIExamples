#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>

namespace msmartcompute {

// Constants for kernel configurations
constexpr int BLOCK_SIZE = 256;
constexpr int WARP_SIZE = 32;
constexpr int MAX_THREADS_PER_BLOCK = 1024;

// Helper function to get grid dimensions
__host__ __forceinline__ dim3 getGridDim(int num_blocks) {
    return dim3((num_blocks + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

// Layer normalization kernel
__global__ void layerNormKernel(
    float* output,
    const float* input,
    const float* weight,
    const float* bias,
    int batch_size,
    int seq_length,
    int hidden_size,
    float epsilon = 1e-5f
) {
    using namespace cooperative_groups;
    auto block = this_thread_block();
    auto warp = tiled_partition<WARP_SIZE>(block);

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int batch_idx = idx / (seq_length * hidden_size);
    const int seq_idx = (idx / hidden_size) % seq_length;
    const int hidden_idx = idx % hidden_size;

    if (batch_idx >= batch_size || seq_idx >= seq_length || hidden_idx >= hidden_size) {
        return;
    }

    // Compute mean
    float sum = 0.0f;
    float sum_sq = 0.0f;
    for (int i = 0; i < hidden_size; i += WARP_SIZE) {
        const int offset = batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i;
        if (i + warp.thread_rank() < hidden_size) {
            const float val = input[offset + warp.thread_rank()];
            sum += val;
            sum_sq += val * val;
        }
    }
    sum = warp.reduce(sum, plus<float>());
    sum_sq = warp.reduce(sum_sq, plus<float>());
    
    const float mean = sum / hidden_size;
    const float var = sum_sq / hidden_size - mean * mean;
    const float inv_std = rsqrtf(var + epsilon);

    // Normalize and scale
    const int out_idx = batch_idx * seq_length * hidden_size + seq_idx * hidden_size + hidden_idx;
    const float normalized = (input[out_idx] - mean) * inv_std;
    output[out_idx] = normalized * weight[hidden_idx] + bias[hidden_idx];
}

// Attention kernel
__global__ void attentionKernel(
    float* output,
    const float* query,
    const float* key,
    const float* value,
    int batch_size,
    int seq_length,
    int num_heads,
    int head_dim,
    float scale
) {
    using namespace cooperative_groups;
    auto block = this_thread_block();
    auto warp = tiled_partition<WARP_SIZE>(block);

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int batch_idx = idx / (seq_length * num_heads * head_dim);
    const int seq_idx = (idx / (num_heads * head_dim)) % seq_length;
    const int head_idx = (idx / head_dim) % num_heads;
    const int dim_idx = idx % head_dim;

    if (batch_idx >= batch_size || seq_idx >= seq_length || 
        head_idx >= num_heads || dim_idx >= head_dim) {
        return;
    }

    // Compute attention scores
    float attention_score = 0.0f;
    for (int k = 0; k < seq_length; ++k) {
        float qk = 0.0f;
        for (int d = 0; d < head_dim; d += WARP_SIZE) {
            if (d + warp.thread_rank() < head_dim) {
                const int q_offset = batch_idx * seq_length * num_heads * head_dim + 
                                   seq_idx * num_heads * head_dim + 
                                   head_idx * head_dim + d + warp.thread_rank();
                const int k_offset = batch_idx * seq_length * num_heads * head_dim + 
                                   k * num_heads * head_dim + 
                                   head_idx * head_dim + d + warp.thread_rank();
                qk += query[q_offset] * key[k_offset];
            }
        }
        qk = warp.reduce(qk, plus<float>());
        attention_score += qk * scale;
    }

    // Apply softmax
    attention_score = __expf(attention_score);
    float sum_exp = 0.0f;
    for (int k = 0; k < seq_length; ++k) {
        sum_exp += attention_score;
    }
    attention_score /= sum_exp;

    // Compute output
    float out_val = 0.0f;
    for (int k = 0; k < seq_length; ++k) {
        for (int d = 0; d < head_dim; d += WARP_SIZE) {
            if (d + warp.thread_rank() < head_dim) {
                const int v_offset = batch_idx * seq_length * num_heads * head_dim + 
                                   k * num_heads * head_dim + 
                                   head_idx * head_dim + d + warp.thread_rank();
                out_val += attention_score * value[v_offset];
            }
        }
    }
    out_val = warp.reduce(out_val, plus<float>());

    const int out_idx = batch_idx * seq_length * num_heads * head_dim + 
                       seq_idx * num_heads * head_dim + 
                       head_idx * head_dim + dim_idx;
    output[out_idx] = out_val;
}

// Feed-forward network kernel
__global__ void ffnKernel(
    float* output,
    const float* input,
    const float* up_weight,
    const float* down_weight,
    int batch_size,
    int seq_length,
    int hidden_size,
    int intermediate_size
) {
    using namespace cooperative_groups;
    auto block = this_thread_block();
    auto warp = tiled_partition<WARP_SIZE>(block);

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int batch_idx = idx / (seq_length * hidden_size);
    const int seq_idx = (idx / hidden_size) % seq_length;
    const int hidden_idx = idx % hidden_size;

    if (batch_idx >= batch_size || seq_idx >= seq_length || hidden_idx >= hidden_size) {
        return;
    }

    // First layer (up projection)
    float intermediate[WARP_SIZE];
    for (int i = 0; i < intermediate_size; i += WARP_SIZE) {
        float sum = 0.0f;
        for (int j = 0; j < hidden_size; ++j) {
            const int in_idx = batch_idx * seq_length * hidden_size + seq_idx * hidden_size + j;
            const int weight_idx = j * intermediate_size + i + warp.thread_rank();
            if (i + warp.thread_rank() < intermediate_size) {
                sum += input[in_idx] * up_weight[weight_idx];
            }
        }
        intermediate[warp.thread_rank()] = sum;
        warp.sync();

        // Apply GELU activation
        if (i + warp.thread_rank() < intermediate_size) {
            intermediate[warp.thread_rank()] = 0.5f * intermediate[warp.thread_rank()] * 
                (1.0f + tanhf(0.797885f * (intermediate[warp.thread_rank()] + 
                0.044715f * intermediate[warp.thread_rank()] * intermediate[warp.thread_rank()])));
        }
    }

    // Second layer (down projection)
    float out_val = 0.0f;
    for (int i = 0; i < intermediate_size; ++i) {
        const int weight_idx = i * hidden_size + hidden_idx;
        out_val += intermediate[i] * down_weight[weight_idx];
    }

    const int out_idx = batch_idx * seq_length * hidden_size + seq_idx * hidden_size + hidden_idx;
    output[out_idx] = out_val;
}

// Host wrapper functions
void launchLayerNorm(
    float* output,
    const float* input,
    const float* weight,
    const float* bias,
    int batch_size,
    int seq_length,
    int hidden_size,
    hipStream_t stream
) {
    const int num_blocks = batch_size * seq_length * hidden_size;
    layerNormKernel<<<getGridDim(num_blocks), BLOCK_SIZE, 0, stream>>>(
        output, input, weight, bias, batch_size, seq_length, hidden_size
    );
}

void launchAttention(
    float* output,
    const float* query,
    const float* key,
    const float* value,
    int batch_size,
    int seq_length,
    int num_heads,
    int head_dim,
    float scale,
    hipStream_t stream
) {
    const int num_blocks = batch_size * seq_length * num_heads * head_dim;
    attentionKernel<<<getGridDim(num_blocks), BLOCK_SIZE, 0, stream>>>(
        output, query, key, value, batch_size, seq_length, num_heads, head_dim, scale
    );
}

void launchFFN(
    float* output,
    const float* input,
    const float* up_weight,
    const float* down_weight,
    int batch_size,
    int seq_length,
    int hidden_size,
    int intermediate_size,
    hipStream_t stream
) {
    const int num_blocks = batch_size * seq_length * hidden_size;
    ffnKernel<<<getGridDim(num_blocks), BLOCK_SIZE, 0, stream>>>(
        output, input, up_weight, down_weight, 
        batch_size, seq_length, hidden_size, intermediate_size
    );
}

} // namespace msmartcompute 