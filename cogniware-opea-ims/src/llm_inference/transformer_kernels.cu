#include "hip/hip_runtime.h"
#include "../include/llm_inference/transformer_kernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <spdlog/spdlog.h>

namespace msmartcompute {
namespace llm_inference {

// Helper functions for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            spdlog::error("CUDA error at {}:{}: {}", __FILE__, __LINE__, hipGetErrorString(error)); \
            return false; \
        } \
    } while(0)

// Attention kernel
__global__ void attentionKernel(
    float* output,
    const float* query,
    const float* key,
    const float* value,
    size_t batch_size,
    size_t seq_length,
    size_t num_heads,
    size_t head_dim,
    float scale
) {
    using namespace cooperative_groups;
    thread_block block = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(block);

    const size_t tid = threadIdx.x;
    const size_t bid = blockIdx.x;
    const size_t head_idx = bid / (batch_size * seq_length);
    const size_t batch_idx = (bid % (batch_size * seq_length)) / seq_length;
    const size_t seq_idx = bid % seq_length;

    // Load query
    float q[32];
    if (tid < head_dim) {
        q[tid] = query[head_idx * head_dim + tid] * scale;
    }
    block.sync();

    // Compute attention scores
    float scores[32];
    for (size_t i = 0; i < seq_length; i += 32) {
        if (i + tid < seq_length) {
            float score = 0.0f;
            for (size_t j = 0; j < head_dim; ++j) {
                score += q[j] * key[i * head_dim + j];
            }
            scores[tid] = score;
        }
    }
    block.sync();

    // Apply softmax
    float max_score = -INFINITY;
    for (size_t i = 0; i < seq_length; i += 32) {
        if (i + tid < seq_length) {
            max_score = max(max_score, scores[tid]);
        }
    }
    max_score = block.reduce(max_score, [](float a, float b) { return max(a, b); });

    float sum_exp = 0.0f;
    for (size_t i = 0; i < seq_length; i += 32) {
        if (i + tid < seq_length) {
            scores[tid] = exp(scores[tid] - max_score);
            sum_exp += scores[tid];
        }
    }
    sum_exp = block.reduce(sum_exp, [](float a, float b) { return a + b; });

    for (size_t i = 0; i < seq_length; i += 32) {
        if (i + tid < seq_length) {
            scores[tid] /= sum_exp;
        }
    }
    block.sync();

    // Compute weighted sum of values
    float output_val[32];
    for (size_t i = 0; i < head_dim; i += 32) {
        if (i + tid < head_dim) {
            float sum = 0.0f;
            for (size_t j = 0; j < seq_length; ++j) {
                sum += scores[j] * value[j * head_dim + i + tid];
            }
            output_val[tid] = sum;
        }
    }
    block.sync();

    // Store output
    if (tid < head_dim) {
        output[head_idx * head_dim + tid] = output_val[tid];
    }
}

// FFN kernel
__global__ void ffnKernel(
    float* output,
    const float* input,
    const float* up_weight,
    const float* down_weight,
    size_t batch_size,
    size_t seq_length,
    size_t hidden_size,
    size_t intermediate_size
) {
    const size_t tid = threadIdx.x;
    const size_t bid = blockIdx.x;
    const size_t batch_idx = bid / seq_length;
    const size_t seq_idx = bid % seq_length;

    // First layer (up projection)
    float intermediate[32];
    for (size_t i = 0; i < intermediate_size; i += 32) {
        if (i + tid < intermediate_size) {
            float sum = 0.0f;
            for (size_t j = 0; j < hidden_size; ++j) {
                sum += input[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + j] * 
                       up_weight[j * intermediate_size + i + tid];
            }
            intermediate[tid] = sum;
        }
    }
    __syncthreads();

    // GELU activation
    for (size_t i = 0; i < intermediate_size; i += 32) {
        if (i + tid < intermediate_size) {
            float x = intermediate[tid];
            intermediate[tid] = 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
        }
    }
    __syncthreads();

    // Second layer (down projection)
    for (size_t i = 0; i < hidden_size; i += 32) {
        if (i + tid < hidden_size) {
            float sum = 0.0f;
            for (size_t j = 0; j < intermediate_size; ++j) {
                sum += intermediate[j] * down_weight[j * hidden_size + i + tid];
            }
            output[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i + tid] = sum;
        }
    }
}

// Layer normalization kernel
__global__ void layerNormKernel(
    float* output,
    const float* input,
    const float* weight,
    const float* bias,
    size_t batch_size,
    size_t seq_length,
    size_t hidden_size
) {
    using namespace cooperative_groups;
    thread_block block = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(block);

    const size_t tid = threadIdx.x;
    const size_t bid = blockIdx.x;
    const size_t batch_idx = bid / seq_length;
    const size_t seq_idx = bid % seq_length;

    // Compute mean
    float sum = 0.0f;
    for (size_t i = tid; i < hidden_size; i += blockDim.x) {
        sum += input[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i];
    }
    sum = block.reduce(sum, [](float a, float b) { return a + b; });
    float mean = sum / hidden_size;

    // Compute variance
    float sq_sum = 0.0f;
    for (size_t i = tid; i < hidden_size; i += blockDim.x) {
        float diff = input[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i] - mean;
        sq_sum += diff * diff;
    }
    sq_sum = block.reduce(sq_sum, [](float a, float b) { return a + b; });
    float variance = sq_sum / hidden_size;

    // Normalize and scale
    for (size_t i = tid; i < hidden_size; i += blockDim.x) {
        float normalized = (input[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i] - mean) / 
                          sqrt(variance + 1e-5f);
        output[batch_idx * seq_length * hidden_size + seq_idx * hidden_size + i] = 
            normalized * weight[i] + bias[i];
    }
}

// Kernel launcher functions
bool launchAttention(
    float* output,
    const float* query,
    const float* key,
    const float* value,
    size_t batch_size,
    size_t seq_length,
    size_t num_heads,
    size_t head_dim,
    float scale,
    hipStream_t stream
) {
    dim3 block(32);
    dim3 grid(batch_size * seq_length * num_heads);
    
    attentionKernel<<<grid, block, 0, stream>>>(
        output, query, key, value,
        batch_size, seq_length, num_heads, head_dim, scale
    );
    
    CUDA_CHECK(hipGetLastError());
    return true;
}

bool launchFFN(
    float* output,
    const float* input,
    const float* up_weight,
    const float* down_weight,
    size_t batch_size,
    size_t seq_length,
    size_t hidden_size,
    size_t intermediate_size,
    hipStream_t stream
) {
    dim3 block(32);
    dim3 grid(batch_size * seq_length);
    
    ffnKernel<<<grid, block, 0, stream>>>(
        output, input, up_weight, down_weight,
        batch_size, seq_length, hidden_size, intermediate_size
    );
    
    CUDA_CHECK(hipGetLastError());
    return true;
}

bool launchLayerNorm(
    float* output,
    const float* input,
    const float* weight,
    const float* bias,
    size_t batch_size,
    size_t seq_length,
    size_t hidden_size,
    hipStream_t stream
) {
    dim3 block(32);
    dim3 grid(batch_size * seq_length);
    
    layerNormKernel<<<grid, block, 0, stream>>>(
        output, input, weight, bias,
        batch_size, seq_length, hidden_size
    );
    
    CUDA_CHECK(hipGetLastError());
    return true;
}

} // namespace llm_inference
} // namespace msmartcompute 